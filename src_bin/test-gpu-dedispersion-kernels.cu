#include "hip/hip_runtime.h"
#include "../include/pirate/internals/GpuDedispersionKernel.hpp"
#include "../include/pirate/internals/ReferenceDedispersionKernel.hpp"
#include "../include/pirate/internals/inlines.hpp"    // pow2()
#include "../include/pirate/internals/utils.hpp"      // integer_log2()
#include "../include/pirate/constants.hpp"            // constants::bytes_per_gpu_cache_line

#include <gputils/Array.hpp>
#include <gputils/cuda_utils.hpp>
#include <gputils/rand_utils.hpp>    // rand_int()
#include <gputils/test_utils.hpp>    // assert_arrays_equal()

using namespace std;
using namespace pirate;
using namespace gputils;


// FIXME delete after de-templating.
template<typename T> struct _is_float32 { };
template<> struct _is_float32<float>   { static constexpr bool value = true; };
template<> struct _is_float32<__half>  { static constexpr bool value = false; };


template<typename T>
struct TestInstance
{
    int rank = 0;
    int ntime = 0;
    int nambient = 1;
    int nbeams = 1;
    int nchunks = 1;
    long row_stride = 0;
    long ambient_stride = 0;
    long beam_stride = 0;
    bool apply_input_residual_lags = false;
    bool is_downsampled_tree = false;
    

    int rand_n(long nmax)
    {
	nmax = min(nmax, 10L);
	nmax = max(nmax, 1L);
	return rand_int(1, nmax+1);
    }

    long rand_stride(long smin)
    {
	int n = max(0L, rand_int(-10,10));
	return smin + 64 * n;  // FIXME 64 -> (128 / sizeof(T))
    }
    
    void randomize()
    {
	const long max_nelts = 30 * 1000 * 1000;

	rank = rand_int(1, 9);
	nchunks = rand_int(1, 10);
	nambient = pow2(rand_int(0,4));
	apply_input_residual_lags = (rand_uniform() < 0.66) ? true : false;
	is_downsampled_tree = (rand_uniform() < 0.5) ? true : false;

	long nelts = pow2(rank) * nchunks * nambient;
	ntime = 64 * rand_n(max_nelts / (64 * nelts));
	nelts *= ntime;
	
	nbeams = rand_n(max_nelts / nelts);
	nelts *= nbeams;
	
	row_stride = rand_stride(ntime);
	ambient_stride = rand_stride(row_stride * pow2(rank));
	beam_stride = rand_stride(ambient_stride * nambient);
    }
    
    
    void run(bool noisy)
    {
	// No real argument checking, but check that everything was initialized.
	assert(rank > 0);
	assert(ntime > 0);
	assert(nambient > 0);
	assert(nbeams > 0);
	assert(nchunks > 0);
	assert(row_stride > 0);
	assert(ambient_stride > 0);
	assert(beam_stride > 0);
	
	if (noisy) {
	    long min_row_stride = ntime;
	    long min_ambient_stride = row_stride * pow2(rank);
	    long min_beam_stride = ambient_stride * nambient;
	    
	    cout << "Test GpuDedispersionKernel\n"
		 << "    dtype = " << gputils::type_name<T>() << "\n"
		 << "    rank = " << rank << "\n"
		 << "    ntime = " << ntime << "\n"
		 << "    nambient = " << nambient << "\n"
		 << "    nbeams = " << nbeams << "\n"
		 << "    nchunks = " << nchunks << "\n"
		 << "    row_stride = " << row_stride << " (minimum: " << min_row_stride << ")\n"
		 << "    ambient_stride = " << ambient_stride << " (minimum: " << min_ambient_stride << ")\n"
		 << "    beam_stride = " << beam_stride << " (minimum: " << min_beam_stride << ")\n"
		 << "    apply_input_residual_lags = " << (apply_input_residual_lags ? "true\n" : "false\n")
		 << "    is_downsampled_tree = " << (is_downsampled_tree ? "true" : "false")
		 << endl;
	}

	ReferenceDedispersionKernel::Params ref_params;
	ref_params.rank = rank;
	ref_params.ntime = ntime;
	ref_params.nambient = nambient;
	ref_params.nbeams = nbeams;
	ref_params.apply_input_residual_lags = apply_input_residual_lags;
	ref_params.is_downsampled_tree = is_downsampled_tree;
	ref_params.nelts_per_segment = xdiv(constants::bytes_per_gpu_cache_line, sizeof(T));  // matches DedispersionConfig::get_nelts_per_segment()

	constexpr bool is_float32 = _is_float32<T>::value;
	typename GpuDedispersionKernel::Params gpu_params;
	gpu_params.dtype = is_float32 ? "float32" : "float16";
	gpu_params.rank = rank;
	gpu_params.nambient = nambient;
	gpu_params.total_beams = nbeams;  // FIXME process in batches
	gpu_params.beams_per_kernel_launch = nbeams;
	gpu_params.ntime = ntime;
	gpu_params.apply_input_residual_lags = apply_input_residual_lags;
	gpu_params.input_is_downsampled_tree = is_downsampled_tree;
	gpu_params.nelts_per_segment = is_float32 ? 32 : 64;

	ReferenceDedispersionKernel ref_kernel(ref_params);
	shared_ptr<GpuDedispersionKernel> gpu_kernel = GpuDedispersionKernel::make(gpu_params);

	Array<T> gpu_iobuf({ nbeams, nambient, pow2(rank), ntime },         // shape
			   { beam_stride, ambient_stride, row_stride, 1 },  // strides
			   af_gpu | af_zero);

	UntypedArray gpu_ubuf;
	if constexpr (is_float32)
	    gpu_ubuf.data_float32 = gpu_iobuf;
	else
	    gpu_ubuf.data_float16 = gpu_iobuf;
	
	for (int ichunk = 0; ichunk < nchunks; ichunk++) {
#if 1
	    // Random chunk gives strongest test.
	    Array<float> ref_chunk({nbeams, nambient, pow2(rank), ntime},
				   { beam_stride, ambient_stride, row_stride, 1 },  // strides
				   af_rhost | af_random);
#else
	    // One-hot chunk is sometimes useful for debugging.
	    // (Note that if nchunks > 0, then the one-hot chunk will be repeated multiple times.)
	    Array<float> ref_chunk({nbeams, nambient, pow2(rank), ntime},
				   { beam_stride, ambient_stride, row_stride, 1 },  // strides
				   af_rhost | af_zero);
	    
	    cout << "   ichunk=" << ichunk << endl;
	    int ibeam = rand_int(0, nbeams);
	    int iamb = rand_int(0, nambient);
	    int irow = rand_int(0, pow2(rank));
	    int it = rand_int(0, ntime);
	    // ibeam=0; iamb=0; irow=0; it=9; // Uncomment if you want a non-random one-hot test
	    cout << "   one-hot chunk: ibeam=" << ibeam << "; iamb=" << iamb << "; irow=" << irow << "; it=" << it << ";" << endl;
	    ref_chunk.at({ibeam,iamb,irow,it}) = 1.0;
#endif

	    // Copy array to GPU before doing reference dedispersion, since reference dedispersion modifies array in-place.
	    gpu_iobuf.fill(ref_chunk.convert_dtype<T>());
	    gpu_kernel->launch(gpu_ubuf, gpu_ubuf, ichunk, 0);
	    CUDA_CALL(hipDeviceSynchronize());
	    Array<float> gpu_output = gpu_iobuf.to_host().template convert_dtype<float> ();
	    
	    ref_kernel.apply(ref_chunk);

#if 0
	    // Sometimes useful for debugging
	    cout << "Printing reference output from chunk " << ichunk << endl;
	    print_array(ref_chunk, {"beam","amb","dmbr","time"});
	    cout << "Printing gpu output from chunk " << ichunk << endl;
	    print_array(gpu_output, {"beam","amb","dmbr","time"});
	    cout << "Printing gpu rstate from chunk " << ichunk << endl;
	    print_array(gpu_rstate.to_host().convert_dtype<float>(), {"beam","amb","ix"});
#endif

	    // FIXME revisit epsilon if we change the normalization of the dedispersion transform.
	    double epsrel = (sizeof(T)==4) ? 1.0e-6 : 0.003;   // float32 vs float16
	    double epsabs = epsrel * pow(1.414, rank);
	    assert_arrays_equal(ref_chunk, gpu_output, "ref", "gpu", {"beam","amb","dmbr","time"}, epsabs, epsrel);
	}

	if (noisy)
	    cout << endl;
    }
};


// -------------------------------------------------------------------------------------------------


int main(int argc, char **argv)
{
    // FIXME switch to 'false' when no longer actively developing
    const bool noisy = true;
    const int niter = 500;

#if 0
    for (int i = 0; i < niter; i++) {
	cout << "Iteration " << i << "/" << niter << "\n\n";
	
	using T = __half;  // float or __half
	TestInstance<T> t;
	t.rank = 7;
	t.ntime = 192;
        t.nambient = 4;
	t.nbeams = 2; 
	t.nchunks = 9;
	t.row_stride = t.ntime + 64;
	t.ambient_stride = t.row_stride * pow2(t.rank) + 64*3;
	t.beam_stride = t.ambient_stride * t.nambient + 64*11;
	t.apply_input_residual_lags = true;
	t.run(noisy);
    }
    return 0;
#endif
    
    for (int i = 0; i < niter; i++) {
	cout << "Iteration " << i << "/" << niter << "\n\n";
	
	TestInstance<__half> th;
	th.randomize();
	th.run(noisy);
	
	TestInstance<float> tf;
	tf.randomize();
	tf.run(noisy);
    }

    cout << "test-gpu-dedispersion-kernels: pass" << endl;
    return 0;
}


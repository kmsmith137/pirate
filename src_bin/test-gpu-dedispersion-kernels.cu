#include "hip/hip_runtime.h"
#include "../include/pirate/internals/GpuDedispersionKernel.hpp"
#include "../include/pirate/internals/ReferenceDedisperser.hpp"
#include "../include/pirate/internals/inlines.hpp"  // pow2()
#include "../include/pirate/internals/utils.hpp"    // integer_log2()

#include <gputils/Array.hpp>
#include <gputils/cuda_utils.hpp>
#include <gputils/rand_utils.hpp>    // rand_int()
#include <gputils/test_utils.hpp>    // assert_arrays_equal()

using namespace std;
using namespace pirate;
using namespace gputils;



template<typename T>
struct ReferenceDedispersionKernel
{
    using RLagType = typename GpuDedispersionKernel<T>::RLagType;

    struct Params {
	int rank = 0;
	int ntime = 0;
	int nambient = 0;
	int nbeams = 0;
	RLagType rlag_type = RLagType::RLagInvalid;
    };

    const Params params;

    shared_ptr<ReferenceTree> tree;
    Array<float> rstate;
    Array<float> scratch;

    // If rlag_type == RLagInput
    shared_ptr<ReferenceLagbuf> rlag_buf;

    
    ReferenceDedispersionKernel(const Params &params_)
	: params(params_)
    {
	// FIXME should have proper argument checking here.
	// Right now, I'm just making sure that everything is initialized.
	
	assert(params.rank > 0);
	assert(params.ntime > 0);
	assert(params.nambient > 0);
	assert(params.nbeams > 0);
	assert((params.rlag_type == RLagType::RLagNone) || (params.rlag_type == RLagType::RLagInput));
	assert(is_power_of_two(params.nambient));

	int B = params.nbeams;
	int A = params.nambient;
	int F = pow2(params.rank);
	int Ar = integer_log2(A);
	
	this->tree = make_shared<ReferenceTree> (params.rank, params.ntime);
	this->rstate = Array<float> ({ B, A, tree->nrstate }, af_uhost | af_zero);
	this->scratch = Array<float> ({ tree->nscratch }, af_uhost | af_zero);
	
	if (params.rlag_type != RLagType::RLagInput)
	    return;

	// Remaining code initializes this->rlag_buf, in case RLagType == RLagInput.
	
	vector<int> rlags(B*A*F);
	constexpr int R = 128 / sizeof(T);

	for (int b = 0; b < B; b++) {
	    for (int a = 0; a < A; a++) {
		// Ambient index represents a bit-reversed DM.
		int dm = bit_reverse_slow(a, Ar);
		
		for (int f = 0; f < F; f++)
		    rlags[b*A*F + a*F + f] = (dm * (F-f-1)) % R;
	    }
	}
	
	this->rlag_buf = make_shared<ReferenceLagbuf> (rlags, params.ntime);
    }

    
    void apply(Array<float> &iobuf) const
    {
	int B = params.nbeams;
	int A = params.nambient;
	int F = pow2(params.rank);

	assert(iobuf.shape_equals({B,A,F,params.ntime}));

	if (params.rlag_type == RLagType::RLagInput) {
	    
	    // FIXME reshape_ref() can fail if A/B/F strides are not compatible.
	    // Some possible solutions:
	    //  - modify ReferenceLagbuf so that 'state' array is passed by caller
	    //  - modify ReferenceLagbuf to allow higher-dimensional data arrays
	    
	    Array<float> iobuf_2d = iobuf.reshape_ref({ B*A*F, params.ntime });
	    rlag_buf->apply_lags(iobuf_2d);
	}

	for (int b = 0; b < B; b++) {
	    for (int a = 0; a < A; a++) {
		Array<float> io_slice = iobuf.slice(0,b).slice(0,a);
		Array<float> rs_slice = rstate.slice(0,b).slice(0,a);

		assert(io_slice.shape_equals({ F, params.ntime }));
		assert(rs_slice.shape_equals({ tree->nrstate }));
		    
		// ReferenceTree::dedisperse(float *arr, int stride, float *rstate, float *scratch)
		tree->dedisperse(io_slice.data, io_slice.strides[0], rs_slice.data, scratch.data);
	    }
	}
    }
};


// -------------------------------------------------------------------------------------------------


template<typename T>
struct TestInstance
{
    using RLagType = typename GpuDedispersionKernel<T>::RLagType;

    int rank = 0;
    int ntime = 0;
    int nambient = 1;
    int nbeams = 1;
    int nchunks = 1;
    long row_stride = 0;
    long ambient_stride = 0;
    long beam_stride = 0;
    RLagType rlag_type = RLagType::RLagInvalid;


    int rand_n(long nmax)
    {
	nmax = min(nmax, 10L);
	nmax = max(nmax, 1L);
	return rand_int(1, nmax+1);
    }

    long rand_stride(long smin)
    {
	int n = max(0L, rand_int(-10,10));
	return smin + 64 * n;  // FIXME 64 -> (128 / sizeof(T))
    }
    
    void randomize()
    {
	const long max_nelts = 30 * 1000 * 1000;
	// const bool is_float32 = (sizeof(T) == 4);

	rank = rand_int(1, 9);
	nchunks = rand_int(1, 10);
	nambient = pow2(rand_int(0,4));
	rlag_type = (rand_uniform() < 0.5) ? RLagType::RLagNone : RLagType::RLagInput;

	long nelts = pow2(rank) * nchunks * nambient;
	ntime = 64 * rand_n(max_nelts / (64 * nelts));
	nelts *= ntime;
	
	nbeams = rand_n(max_nelts / nelts);
	nelts *= nbeams;
	
	row_stride = rand_stride(ntime);
	ambient_stride = rand_stride(row_stride * pow2(rank));
	beam_stride = rand_stride(ambient_stride * nambient);
    }
    
    
    void run(bool noisy)
    {
	// No real argument checking, but check that everything was initialized.
	assert(rank > 0);
	assert(ntime > 0);
	assert(nambient > 0);
	assert(nbeams > 0);
	assert(nchunks > 0);
	assert(row_stride > 0);
	assert(ambient_stride > 0);
	assert(beam_stride > 0);
	
	if (noisy) {
	    long min_row_stride = ntime;
	    long min_ambient_stride = row_stride * pow2(rank);
	    long min_beam_stride = ambient_stride * nambient;
	    
	    cout << "Test GpuDedispersionKernel\n"
		 << "    dtype = " << gputils::type_name<T>() << "\n"
		 << "    rank = " << rank << "\n"
		 << "    ntime = " << ntime << "\n"
		 << "    nambient = " << nambient << "\n"
		 << "    nbeams = " << nbeams << "\n"
		 << "    nchunks = " << nchunks << "\n"
		 << "    row_stride = " << row_stride << " (minimum: " << min_row_stride << ")\n"
		 << "    ambient_stride = " << ambient_stride << " (minimum: " << min_ambient_stride << ")\n"
		 << "    beam_stride = " << beam_stride << " (minimum: " << min_beam_stride << ")\n"
		 << "    rlag_type = " << GpuDedispersionKernel<T>::rlag_str(rlag_type)
		 << endl;
	}

	using RefParams = typename ReferenceDedispersionKernel<T>::Params;
	RefParams ref_params;
	ref_params.rank = rank;
	ref_params.ntime = ntime;
	ref_params.nambient = nambient;
	ref_params.nbeams = nbeams;
	ref_params.rlag_type = rlag_type;

	ReferenceDedispersionKernel<T> ref_kernel(ref_params);

	shared_ptr<GpuDedispersionKernel<T>> gpu_kernel = GpuDedispersionKernel<T>::make(rank, rlag_type);

	if (noisy)
	    gpu_kernel->print(cout, 4);  // indent=4

	Array<T> gpu_iobuf({ nbeams, nambient, pow2(rank), ntime },         // shape
			   { beam_stride, ambient_stride, row_stride, 1 },  // strides
			   af_gpu | af_zero);
	
	Array<T> gpu_rstate({ nbeams, nambient, gpu_kernel->params.state_nelts_per_small_tree },
			    af_gpu | af_zero);
	
	for (int ichunk = 0; ichunk < nchunks; ichunk++) {
#if 1
	    // Random chunk gives strongest test.
	    Array<float> ref_chunk({nbeams, nambient, pow2(rank), ntime}, af_rhost | af_random);
#else
	    // One-hot chunk is sometimes useful for debugging.
	    // (Note that if nchunks > 0, then the one-hot chunk will be repeated multiple times.)
	    Array<float> ref_chunk({nbeams, nambient, pow2(rank), ntime}, af_rhost | af_zero);
	    cout << "   ichunk=" << ichunk << endl;
	    int ibeam = rand_int(0, nbeams);
	    int iamb = rand_int(0, nambient);
	    int irow = rand_int(0, pow2(rank));
	    int it = rand_int(0, ntime);
	    // ibeam=0; iamb=0; irow=0; it=9; // Uncomment if you want a non-random one-hot test
	    cout << "   one-hot chunk: ibeam=" << ibeam << "; iamb=" << iamb << "; irow=" << irow << "; it=" << it << ";" << endl;
	    ref_chunk.at({ibeam,iamb,irow,it}) = 1.0;
#endif

	    // Copy array to GPU before doing reference dedispersion, since reference dedispersion modifies array in-place.
	    gpu_iobuf.fill(ref_chunk.convert_dtype<T>());
	    gpu_kernel->launch(gpu_iobuf, gpu_rstate);
	    CUDA_CALL(hipDeviceSynchronize());
	    Array<float> gpu_output = gpu_iobuf.to_host().template convert_dtype<float> ();
	    
	    ref_kernel.apply(ref_chunk);

#if 0
	    // Sometimes useful for debugging
	    cout << "Printing reference output from chunk " << ichunk << endl;
	    print_array(ref_chunk, {"beam","amb","dmbr","time"});
	    cout << "Printing gpu output from chunk " << ichunk << endl;
	    print_array(gpu_output, {"beam","amb","dmbr","time"});
	    cout << "Printing gpu rstate from chunk " << ichunk << endl;
	    print_array(gpu_rstate.to_host().convert_dtype<float>(), {"beam","amb","ix"});
#endif

	    // FIXME revisit epsilon if we change the normalization of the dedispersion transform.
	    double epsrel = (sizeof(T)==4) ? 1.0e-6 : 0.003;   // float32 vs float16
	    double epsabs = epsrel * pow(1.414, rank);
	    assert_arrays_equal(ref_chunk, gpu_output, "ref", "gpu", {"beam","amb","dmbr","time"}, epsabs, epsrel);
	}

	if (noisy)
	    cout << endl;
    }
};


// -------------------------------------------------------------------------------------------------


int main(int argc, char **argv)
{
    // FIXME switch to 'false' when no longer actively developing
    const bool noisy = true;
    const int niter = 500;

#if 0
    for (int i = 0; i < niter; i++) {
	cout << "Iteration " << i << "/" << niter << "\n\n";
	
	using T = __half;  // float or __half
	TestInstance<T> t;
	t.rank = 7;
	t.ntime = 192;
        t.nambient = 4;
	t.nbeams = 2; 
	t.nchunks = 9;
	t.row_stride = t.ntime + 64;
	t.ambient_stride = t.row_stride * pow2(t.rank) + 64*3;
	t.beam_stride = t.ambient_stride * t.nambient + 64*11;
	t.rlag_type = GpuDedispersionKernel<T>::RLagType::RLagInput;
	t.run(noisy);
    }
    return 0;
#endif
    
    for (int i = 0; i < niter; i++) {
	cout << "Iteration " << i << "/" << niter << "\n\n";
	
	TestInstance<__half> th;
	th.randomize();
	th.run(noisy);
	
	TestInstance<float> tf;
	tf.randomize();
	tf.run(noisy);
    }

    cout << "test-gpu-dedispersion-kernels: pass" << endl;
    return 0;
}


#include "hip/hip_runtime.h"
#include "../include/pirate/DedispersionPlan.hpp"
#include "../include/pirate/DedispersionConfig.hpp"
#include "../include/pirate/internals/ReferenceDedisperser.hpp"
#include "../include/pirate/internals/inlines.hpp"  // pow2()
#include "../include/pirate/internals/utils.hpp"    // make_chord_dedispersion_config()

#include <gputils/Array.hpp>
#include <gputils/rand_utils.hpp>
#include <gputils/test_utils.hpp>    // assert_arrays_equal()

using namespace std;
using namespace gputils;
using namespace pirate;


// -------------------------------------------------------------------------------------------------


static void test_reference_dedispersion(const DedispersionConfig &config, int soph1, int soph2, int nchunks, bool noisy=false)
{
    if (noisy) {
	cout << "test_reference_dedispersion(soph1=" << soph1
	     << ", soph2=" << soph2 << ", nchunks=" << nchunks << ")" << endl;
	config.print(cout, 4);
    }
    
    shared_ptr<DedispersionPlan> plan = make_shared<DedispersionPlan> (config);
    ReferenceDedisperser rdd1(plan, soph1);
    ReferenceDedisperser rdd2(plan, soph2);

    assert(rdd1.output_ntrees == rdd2.output_ntrees);
    
    int nfreq = pow2(config.tree_rank);
    int nt_chunk = config.time_samples_per_chunk;
    int ntrees = rdd1.output_ntrees;
    
    for (int c = 0; c < nchunks; c++) {
	if (noisy)
	    cout << "    chunk " << c << "/" << nchunks << endl;
	
	Array<float> arr({nfreq,nt_chunk}, af_uhost | af_random);
	// Array<float> arr({nfreq,nt_chunk}, af_uhost | af_zero);
	// arr.at({0,0}) = 1.0;
	
	rdd1.dedisperse(arr);
	rdd2.dedisperse(arr);

	for (int itree = 0; itree < ntrees; itree++) {
	    const Array<float> &arr1 = rdd1.output_arrays.at(itree);
	    const Array<float> &arr2 = rdd2.output_arrays.at(itree);
	    assert_arrays_equal(arr1, arr2, "A", "B", {"dm_brev","t"});
	}
    }
}


// -------------------------------------------------------------------------------------------------


static void run_random_small_configs(int niter)
{
    for (int iter = 0; iter < niter; iter++) {
	cout << "\n    *** Running random small config " << iter << "/" << niter << " ***\n" << endl;
	
	auto config = DedispersionConfig::make_random();
	config.planner_verbosity = 1;

	int max_nt = 8192;
	assert(config.time_samples_per_chunk <= max_nt);
	
	int max_nchunks = max_nt / config.time_samples_per_chunk;  // round down
	int nchunks = gputils::rand_int(1, max_nchunks+1);
	
	test_reference_dedispersion(config, 0, 3, nchunks, true);  // noisy=true
    }
}


int main(int argc, char **argv)
{
    if (argc == 1) {
	const int niter = 100;
	
	cout << "No command-line arguments were specified; running "
	     << niter << " randomly generated 'small' configs" << endl;
	
	run_random_small_configs(niter);
	
	cout << "\nThis concludes our test of " << niter << " randomly generated 'small' configs.\n"
	     << "To run a long test, specify a config on the command line, e.g.\n"
	     << "   ./bin/test-reference-dedisperser configs/dedispersion/chord_zen3/chord_zen3_int8_float16.yml\n";
    }

    for (int iarg = 1; iarg < argc; iarg++) {
	auto config = DedispersionConfig::from_yaml(argv[iarg]);
    
	int nt_tot = 1024 * 1024;  // FIXME promote to command-line arg?
	int nchunks = xdiv(nt_tot, config.time_samples_per_chunk);
	test_reference_dedispersion(config, 0, 3, nchunks, true);  // noisy=true
    }
    
    cout << "\ntest-reference-dedisperser: pass" << endl;
    return 0;
}


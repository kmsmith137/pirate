#include "hip/hip_runtime.h"
#include <iostream>
#include <gputils/Array.hpp>
#include <gputils/CudaStreamPool.hpp>

#include "../include/pirate/internals/inlines.hpp"  // pow2()
#include "../include/pirate/internals/GpuDedispersionKernel.hpp"

using namespace std;
using namespace gputils;
using namespace pirate;


template<typename T>
static void time_gpu_dedispersion_kernel(int rank, typename GpuDedispersionKernel<T>::RLagType rlag_type)
{
    shared_ptr<GpuDedispersionKernel<T>> kernel = GpuDedispersionKernel<T>::make(rank, rlag_type);
    
    long nstreams = 1;
    long ncallbacks = 10;
    long nambient = 256;
    long nbeams = pow2(12-rank) / sizeof(T);
    long ntime = 2048;
    long niter = 20;

    Array<T> iobuf({nstreams, nbeams, nambient, pow2(rank), ntime}, af_zero | af_gpu);
    Array<T> rstate({nstreams, nbeams, nambient, kernel->params.state_nelts_per_small_tree}, af_zero | af_gpu);
    
    long iobuf_bytes_per_stream = nbeams * nambient * pow2(rank) * ntime * sizeof(T);
    long rstate_bytes_per_stream = nbeams * nambient * kernel->params.state_nelts_per_small_tree * sizeof(T);
    double gmem_gb = 2.0e-9 * niter * (iobuf_bytes_per_stream + rstate_bytes_per_stream);  // factor 2 from read+write
    
    auto callback = [&](const CudaStreamPool &pool, hipStream_t stream, int istream)
        {
	    Array<T> iobuf_s = iobuf.slice(0, istream);
	    Array<T> rstate_s = rstate.slice(0, istream);

	    for (int i = 0; i < niter; i++)
		kernel->launch(iobuf_s, rstate_s);
	};
    
    stringstream kernel_name;
    kernel_name << "dedisperse(" << gputils::type_name<T>() << ", rank=" << rank
		<< ", " << GpuDedispersionKernel<T>::rlag_str(rlag_type) << ")";
    
    CudaStreamPool pool(callback, ncallbacks, nstreams, kernel_name.str());
    pool.monitor_throughput("global memory (GB/s)", gmem_gb);
    pool.run();
}


int main(int argc, char **argv)
{
    for (int rank = 1; rank <= 8; rank++) {
	time_gpu_dedispersion_kernel<float> (rank, GpuDedispersionKernel<float>::RLagNone);
	time_gpu_dedispersion_kernel<float> (rank, GpuDedispersionKernel<float>::RLagInput);
	time_gpu_dedispersion_kernel<__half> (rank, GpuDedispersionKernel<__half>::RLagNone);
	time_gpu_dedispersion_kernel<__half> (rank, GpuDedispersionKernel<__half>::RLagInput);
    }
    
    return 0;
}

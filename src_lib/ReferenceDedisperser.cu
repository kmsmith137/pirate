#include "hip/hip_runtime.h"
#include "../include/pirate/internals/ReferenceDedisperser.hpp"

#include "../include/pirate/constants.hpp"
#include "../include/pirate/internals/inlines.hpp"
#include "../include/pirate/internals/utils.hpp"


using namespace std;
using namespace gputils;

namespace pirate {
#if 0
}  // editor auto-indent
#endif


// -------------------------------------------------------------------------------------------------
//
// Stage0Buffers


// Helper class used in ReferenceDedisperser1, ReferenceDedisperser2.
struct Stage0Buffers
{
    shared_ptr<DedispersionPlan> plan;
    
    long nds = 0;                  // same as plan->stage0_trees.size()
    long nseg = 0;                 // same as plan->stage0_total_segments_per_beam
    long nelts_per_segment = 0;    // same as plan->nelts_per_segment
    long beams_per_batch = 0;      // same as plan->config.beams_per_batch
    long total_beams = 0;          // same as plan->config.beams_per_gpu
    long nbatches = 0;             // same as (total_beams / beams_per_batch)
    
    Array<float> flat_buf;         // shape (beams_per_batch, nseg * nelts_per_segment)
    vector<Array<float>> dd_bufs;  // length nds, inner shape is (beams_per_batch, pow2(st0.rank), nt_ds)
    vector<Array<float>> ds_bufs;  // length (nds-1), same as dd_bufs[1:]

    vector<shared_ptr<ReferenceLaggedDownsampler>> lds_kernels;   // length (nbatches)
    vector<shared_ptr<ReferenceDedispersionKernel>> dd_kernels;   // length (nds)

    Stage0Buffers(const shared_ptr<DedispersionPlan> &plan_) : plan(plan_)
    {
	this->nds = plan->config.num_downsampling_levels;
	this->nseg = plan->stage0_total_segments_per_beam;
	this->nelts_per_segment = plan->nelts_per_segment;
	this->beams_per_batch = plan->config.beams_per_batch;
	this->total_beams = plan->config.beams_per_gpu;
	this->nbatches = xdiv(total_beams, beams_per_batch);

	this->dd_bufs.resize(nds);
	this->ds_bufs.resize(nds-1);
	this->lds_kernels.resize(nbatches);
	this->dd_kernels.resize(nds);
	
	// Allocate buffers.

	this->flat_buf = Array<float> ({beams_per_batch, nseg * nelts_per_segment}, af_uhost | af_zero);
	long pos = 0;
	
	for (long ids = 0; ids < nds; ids++) {
	    const DedispersionPlan::Stage0Tree &st0 = plan->stage0_trees.at(ids);
	    assert(pos == st0.base_segment * nelts_per_segment);
	    
	    long nelts = st0.segments_per_beam * nelts_per_segment;
	    long rank = st0.rank0 + st0.rank1;
	    long nt_ds = st0.nt_ds;

	    assert(nelts == pow2(rank) * nt_ds);
	    Array<float> view = flat_buf.slice(1, pos, pos+nelts);
	    view = view.reshape_ref({ beams_per_batch, pow2(rank), nt_ds });

	    dd_bufs.at(ids) = view;
	    
	    if (ids > 0)
		ds_bufs.at(ids-1) = view;
	    
	    pos += nelts;
	}
	
	// LaggedDownsampler kernels.

	if (nds > 1) {
	    ReferenceLaggedDownsampler::Params ld_params;
	    ld_params.small_input_rank = plan->stage0_trees.at(1).rank0 + 1;
	    ld_params.large_input_rank = plan->config.tree_rank;
	    ld_params.num_downsampling_levels = nds - 1;   // note (-1) here!
	    ld_params.nbeams = beams_per_batch;
	    ld_params.ntime = plan->config.time_samples_per_chunk;

	    for (long b = 0; b < nbatches; b++)
		this->lds_kernels.at(b) = make_shared<ReferenceLaggedDownsampler> (ld_params);
	}

	// Dedispersion kernels.

	for (long ids = 0; ids < nds; ids++) {
	    const DedispersionPlan::Stage0Tree &st0 = plan->stage0_trees.at(ids);
	    
	    ReferenceDedispersionKernel::Params params;
	    params.dtype = plan->config.dtype;
	    params.rank = st0.rank0;
	    params.nambient = pow2(st0.rank1);
	    params.total_beams = plan->config.beams_per_gpu;
	    params.beams_per_kernel_launch = plan->config.beams_per_batch;
	    params.ntime = st0.nt_ds;
	    params.input_is_ringbuf = false;
	    params.output_is_ringbuf = false;
	    params.apply_input_residual_lags = false;
	    params.input_is_downsampled_tree = (ids > 0);
	    params.nelts_per_segment = plan->nelts_per_segment;
	    
	    dd_kernels.at(ids) = make_shared<ReferenceDedispersionKernel> (params);
	}
    }

    void apply_lagged_downsampler(long ibeam)
    {
	if (nds > 1) {
	    long b = xdiv(ibeam, beams_per_batch);
	    lds_kernels.at(b)->apply(dd_bufs.at(0), ds_bufs);
	}
    }
    
    void apply_dedispersion_kernels(long itime, long ibeam)
    {
	for (long ids = 0; ids < nds; ids++) {
	    const DedispersionPlan::Stage0Tree &st0 = plan->stage0_trees.at(ids);
	    Array<float> buf = dd_bufs.at(ids);
	    buf = buf.reshape_ref({beams_per_batch, pow2(st0.rank1), pow2(st0.rank0), st0.nt_ds});  // shape (1, 2^rank1, 2^rank0, nt_ds)
	    dd_kernels.at(ids)->apply(buf, buf, itime, ibeam);
	}
    }
};


// -------------------------------------------------------------------------------------------------
//
// Stage1Buffers


// Helper class used in ReferenceDedisperser1, ReferenceDedisperser2.
struct Stage1Buffers
{
    shared_ptr<DedispersionPlan> plan;

    long nout = 0;                // same as plan->stage1_trees.size()
    long nseg = 0;                // same as plan->stage1_total_segments_per_beam
    long nelts_per_segment = 0;   // same as plan->nelts_per_segment
    long beams_per_batch = 0;     // same as plan->config.beams_per_batch
    long total_beams = 0;         // same as plan->config.beams_per_gpu
    long nbatches = 0;            // same as (total_beams / beams_per_batch)

    Array<float> flat_buf;         // shape (beams_per_batch, nseg * nelts_per_segment)
    vector<Array<float>> dd_bufs;  // length nout, inner shape is (beams_per_batch, pow2(st1.rank), nt_ds)
    vector<shared_ptr<ReferenceDedispersionKernel>> dd_kernels;   // length (nout)

    
    Stage1Buffers(const shared_ptr<DedispersionPlan> &plan_) : plan(plan_)
    {
	this->nout = plan->stage1_trees.size();
	this->nseg = plan->stage1_total_segments_per_beam;
	this->nelts_per_segment = plan->nelts_per_segment;
	this->beams_per_batch = plan->config.beams_per_batch;
	this->total_beams = plan->config.beams_per_gpu;
	this->nbatches = xdiv(total_beams, beams_per_batch);

	this->dd_bufs.resize(nout);
	this->dd_kernels.resize(nout);
	
	// Allocate buffers.

	this->flat_buf = Array<float> ({beams_per_batch, nseg * nelts_per_segment}, af_uhost | af_zero);
	long pos = 0;

	for (long iout = 0; iout < nout; iout++) {
	    const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	    assert(pos == st1.base_segment * nelts_per_segment);
	    
	    long nelts = st1.segments_per_beam * nelts_per_segment;
	    long rank = st1.rank0 + st1.rank1_trigger;
	    long nt_ds = st1.nt_ds;

	    assert(nelts == pow2(rank) * nt_ds);
	    Array<float> view = flat_buf.slice(1, pos, pos+nelts);
	    view = view.reshape_ref({ beams_per_batch, pow2(rank), nt_ds });

	    dd_bufs.at(iout) = view;
	    pos += nelts;
	}

	// Dedispersion kernels.

	for (long iout = 0; iout < nout; iout++) {
	    const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	    
	    ReferenceDedispersionKernel::Params params;		
	    params.dtype = plan->config.dtype;
	    params.rank = st1.rank1_trigger;
	    params.nambient = pow2(st1.rank0);
	    params.total_beams = total_beams;
	    params.beams_per_kernel_launch = beams_per_batch;
	    params.ntime = st1.nt_ds;
	    params.input_is_ringbuf = false;
	    params.output_is_ringbuf = false;
	    params.apply_input_residual_lags = true;
	    params.input_is_downsampled_tree = (st1.ds_level > 0);
	    params.nelts_per_segment = plan->nelts_per_segment;
	    
	    dd_kernels.at(iout) = make_shared<ReferenceDedispersionKernel> (params);
	}
    }

    void apply_dedispersion_kernels(long itime, long ibeam)
    {
	for (int iout = 0; iout < nout; iout++) {
	    const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	    long rank0 = st1.rank0;
	    long rank1 = st1.rank1_trigger;

	    Array<float> buf = dd_bufs.at(iout);  // shape (beams_per_batch, 2^(rank0+rank1), nt_ds)
	    buf = buf.reshape_ref({beams_per_batch, pow2(rank1), pow2(rank0), st1.nt_ds});
	    buf = buf.transpose({0,2,1,3});       // shape (beams_per_batch, 2^rank0, 2^rank1, nt_ds)
	    dd_kernels.at(iout)->apply(buf, buf, itime, ibeam);
	}
    }
};


// -------------------------------------------------------------------------------------------------
//
// ReferenceDedisperserBase


// Helper for ReferenceDedisperserBase constructor.
// Prevents constructor from segfaulting, if invoked with empty shared_ptr.
static DedispersionPlan *deref(const shared_ptr<DedispersionPlan> &p)
{
    if (!p)
	throw runtime_error("ReferenceDedisperser constructor called with empty shared_ptr");
    return p.get();
}


ReferenceDedisperserBase::ReferenceDedisperserBase(const shared_ptr<DedispersionPlan> &plan_, int sophistication_) :
    plan(plan_),
    config(deref(plan_)->config),
    sophistication(sophistication_)
{
    this->config_rank = config.tree_rank;
    this->config_ntime = config.time_samples_per_chunk;
    this->total_beams = config.beams_per_gpu;
    this->beams_per_batch = config.beams_per_batch;
    this->nbatches = xdiv(total_beams, beams_per_batch);

    this->nds = plan->stage0_trees.size();
    this->nout = plan->stage1_trees.size();
    this->nelts_per_segment = plan->nelts_per_segment;

    // Note: 'input_array' and 'output_arrays' are members of ReferenceDedisperserBase,
    // but are initialized by the subclass constructor.
}


void ReferenceDedisperserBase::check_iobuf_shapes()
{
    assert(input_array.shape_equals({ beams_per_batch, pow2(config_rank), config_ntime }));
    assert(output_arrays.size() == nout);

    for (long iout = 0; iout < nout; iout++) {
	const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	int rank = st1.rank0 + st1.rank1_trigger;
	assert(output_arrays.at(iout).shape_equals({ beams_per_batch, pow2(rank), st1.nt_ds }));
    }
}

void ReferenceDedisperserBase::dedisperse(long itime, long ibeam)
{
    assert(itime == expected_itime);
    assert(ibeam == expected_ibeam);
    
    this->_dedisperse(itime, ibeam);

    expected_ibeam += beams_per_batch;

    if (expected_ibeam >= total_beams) {
	expected_itime++;
	expected_ibeam = 0;
    }
}


// Note: ReferenceDedisperserBase::make() is defined at the end of the file.


// -------------------------------------------------------------------------------------------------
//
// Sophistication == 0:
//
//   - Uses one-stage dedispersion instead of two stages.
//   - In downsampled trees, compute twice as many DMs as necessary, then drop the bottom half.
//   - Each early trigger is computed in an independent tree, by disregarding some input channels.


struct ReferenceDedisperser0 : public ReferenceDedisperserBase
{
    ReferenceDedisperser0(const shared_ptr<DedispersionPlan> &plan);

    virtual void _dedisperse(long itime, long ibeam) override;

    // Step 1: downsample input array (straightforward downsample, not "lagged" downsample!)
    // Outer length is nds, inner shape is (beams_per_batch, 2^config_rank, input_nt / pow2(ids)).
    
    vector<Array<float>> downsampled_inputs;

    // Step 2: copy from 'downsampled_inputs' to 'dedispersion_buffers'.
    // In downsampled trees, we compute twice as many DMs as necessary, then drop the bottom half.
    // Each early trigger is computed in an independent tree, by disregarding some input channels.
    // Outer vector length is nout, inner shape is (beams_per_batch, 2^weird_rank, input_nt / pow2(ids)).
    //   where weird_rank = rank0 + rank1_trigger + (is_downsampled ? 1 : 0)
    
    vector<Array<float>> dedispersion_buffers;

    // Step 3: apply tree dedispersion (one-stage, not two-stage).
    // Vector length is (nbatches * nout).
    // Inner shape is (beams_per_batch, 2^weird_rank, input_nt / pow2(ids)).
    
    vector<shared_ptr<ReferenceTree>> trees;

    // Step 4: copy from 'dedispersion_buffers' to 'output_arrays'.
    // In downsampled trees, we compute twice as many DMs as necessary, then copy the bottom half.
    // Reminder: 'output_arrays' is a member of ReferenceDedisperserBase.
};


ReferenceDedisperser0::ReferenceDedisperser0(const shared_ptr<DedispersionPlan> &plan_) :
    ReferenceDedisperserBase(plan_, 0)
{    
    this->downsampled_inputs.resize(nds);
    this->dedispersion_buffers.resize(nout);
    this->trees.resize(nbatches * nout);    
    this->output_arrays.resize(nout);

    for (int ids = 0; ids < nds; ids++) {
	long nt_ds = xdiv(config_ntime, pow2(ids));
	downsampled_inputs.at(ids) = Array<float> ({beams_per_batch, pow2(config_rank), nt_ds}, af_uhost | af_zero);
    }
    
    for (int iout = 0; iout < nout; iout++) {
	const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	long ids = st1.ds_level;
	long out_rank = st1.rank0 + st1.rank1_trigger;
	long weird_rank = out_rank + (ids ? 1 : 0);
	
	this->dedispersion_buffers.at(iout) = Array<float> ({beams_per_batch, pow2(weird_rank), st1.nt_ds}, af_uhost | af_zero);
	this->output_arrays.at(iout) = Array<float>({beams_per_batch, pow2(out_rank), st1.nt_ds}, af_uhost | af_zero);

	for (int batch = 0; batch < nbatches; batch++)
	    this->trees.at(batch*nout + iout) = ReferenceTree::make({ beams_per_batch, pow2(weird_rank), st1.nt_ds });
    }
    
    // Reminder: 'input_array' and 'output_arrays' are members of ReferenceDedisperserBase,
    // but are initialized by the subclass constructor.

    this->input_array = downsampled_inputs.at(0);   // alias
    this->check_iobuf_shapes();
}


// virtual override
void ReferenceDedisperser0::_dedisperse(long itime, long ibeam)
{
    for (int ids = 1; ids < nds; ids++) {
	
	// Step 1: downsample input array (straightforward downsample, not "lagged" downsample).
	// Outer length is nds, inner shape is (beams_per_batch, 2^config_rank, input_nt / pow2(ids)).
	// Reminder: 'input_array' is an alias for downsampled_inputs[0].

	Array<float> src = downsampled_inputs.at(ids-1);
	Array<float> dst = downsampled_inputs.at(ids);

	// FIXME reference_downsample_time() should operate on N-dimensional array.
	for (long b = 0; b < beams_per_batch; b++) {
	    Array<float> src2 = src.slice(0,b);
	    Array<float> dst2 = dst.slice(0,b);
	    reference_downsample_time(src2, dst2, false);  // normalize=false, i.e. no factor 0.5
	}
    }

    for (int iout = 0; iout < nout; iout++) {
	const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	long ids = st1.ds_level;
	long weird_rank = st1.rank0 + st1.rank1_trigger + (ids ? 1 : 0);
	
	Array<float> in = downsampled_inputs.at(ids).slice(1, 0, pow2(weird_rank));
	Array<float> dd = dedispersion_buffers.at(iout);
	Array<float> out = output_arrays.at(iout);
	
	// Step 2: copy from 'downsampled_inputs' to 'dedispersion_buffers'.
	
	dd.fill(in);

	// Step 3: apply tree dedispersion (one-stage, not two-stage).
	// Vector length is (nbatches * nout).
	
	long batch = xdiv(ibeam, beams_per_batch);
	auto tree = trees.at(batch*nout + iout);
	tree->dedisperse(dd);
	
	// Step 4: copy from 'dedispersion_buffers' to 'output_arrays'.
	// In downsampled trees, we compute twice as many DMs as necessary, then copy the bottom half.
	
	if (ids == 0)
	    out.fill(dd);
	else {
	    // FIXME refence_extract_odd_channels() should operate on N-dimensional array.
	    // reference_extract_odd_channels(dd, out);
	    for (long b = 0; b < beams_per_batch; b++) {
		Array<float> src2 = dd.slice(0,b);
		Array<float> dst2 = out.slice(0,b);
		reference_extract_odd_channels(src2, dst2);
	    }
	}
    }
}


// -------------------------------------------------------------------------------------------------
//
// Sophistication == 1:
//
//   - Uses same two-stage tree/lag structure as plan.
//   - Lags are split into segments + residuals, but not further split into chunks.
//   - Lags are applied with a per-tree ReferenceLagbuf, rather than using ring/staging buffers.


struct ReferenceDedisperser1 : public ReferenceDedisperserBase
{
    ReferenceDedisperser1(const shared_ptr<DedispersionPlan> &plan_);

    // Step 1: run LaggedDownsampler.
    // Step 2: run stage0 dedispersion kernels 
    Stage0Buffers stage0_buffers;

    // Step 3: copy stage0 -> stage1
    Stage1Buffers stage1_buffers;
    
    // Step 4: apply lags
    // Step 5: run stage1 dedispersion kernels.
    vector<shared_ptr<ReferenceLagbuf>> stage1_lagbufs;  // length (nbatches * nout)
    
    virtual void _dedisperse(long itime, long ibeam) override;
};


ReferenceDedisperser1::ReferenceDedisperser1(const shared_ptr<DedispersionPlan> &plan_) :
    ReferenceDedisperserBase(plan_, 1),
    stage0_buffers(plan_),
    stage1_buffers(plan_)
{
    long S = nelts_per_segment;
    
    // Reminder: 'input_array' and 'output_arrays' are members of ReferenceDedisperserBase,
    // but are initialized by the subclass constructor.

    this->input_array = stage0_buffers.dd_bufs.at(0);  // alias
    this->output_arrays = stage1_buffers.dd_bufs;      // alias
    this->check_iobuf_shapes();

    this->stage1_lagbufs.resize(nbatches * nout);

    for (long iout = 0; iout < nout; iout++) {
	const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	int rank0 = st1.rank0;
	int rank1 = st1.rank1_trigger;
	int nchan = pow2(rank0+rank1);
	bool is_downsampled = (st1.ds_level > 0);

	Array<int> lags({beams_per_batch, nchan}, af_uhost);

	for (long i1 = 0; i1 < pow2(rank1); i1++) {
	    for (long i0 = 0; i0 < pow2(rank0); i0++) {
		int row = i1 * pow2(rank0) + i0;
		int lag = rb_lag(i1, i0, rank0, rank1, is_downsampled);
		int segment_lag = lag / S;   // round down

		for (long b = 0; b < beams_per_batch; b++)
		    lags.data[b*nchan + row] = segment_lag * S;
	    }
	}

	for (long b = 0; b < nbatches; b++)
	    stage1_lagbufs.at(b*nout + iout) = make_shared<ReferenceLagbuf> (lags, st1.nt_ds);
    }
}


// virtual override
void ReferenceDedisperser1::_dedisperse(long itime, long ibeam)
{
    // Step 1: run LaggedDownsampler.
    // Step 2: run stage0 dedispersion kernels.
    this->stage0_buffers.apply_lagged_downsampler(ibeam);    
    this->stage0_buffers.apply_dedispersion_kernels(itime, ibeam);

    for (int iout = 0; iout < nout; iout++) {
	const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	long rank0 = st1.rank0;
	long rank1 = st1.rank1_trigger;

	// Step 3: copy stage0 -> stage1
	
	Array<float> src = stage0_buffers.dd_bufs.at(st1.ds_level);  // shape (beams_per_batch, 2^rank_ambient, nt_ds)
	src = src.slice(1, 0, pow2(rank0+rank1));                    // shape (beams_per_batch, 2^rank, nt_ds)

	Array<float> dst = stage1_buffers.dd_bufs.at(iout);
	dst.fill(src);

	// Step 4: apply lags
	
	long b = xdiv(ibeam, beams_per_batch);
	auto lagbuf = stage1_lagbufs.at(b*nout + iout);
	lagbuf->apply_lags(dst);
    }

    // Step 5: run stage1 dedispersion kernels
    this->stage1_buffers.apply_dedispersion_kernels(itime, ibeam);
}


// -------------------------------------------------------------------------------------------------
//
// ReferenceDedisperser2: as close to the GPU implementation as possible.


struct ReferenceDedisperser2 : public ReferenceDedisperserBase
{
    ReferenceDedisperser2(const std::shared_ptr<DedispersionPlan> &plan);

    virtual void _dedisperse(long itime, long ibeam) override;
    
    // Step 1: run LaggedDownsampler.
    // Step 2: run stage0 dedispersion kernels.
    Stage0Buffers stage0_buffers;
    
    // Step 3: copy stage0 -> ringbuf.
    Array<float> gpu_ringbuf;

    // Step 4: copy ringbuf -> stage1.
    // Step 5: run stage1 dedispersion kernels.
    Stage1Buffers stage1_buffers;
};


ReferenceDedisperser2::ReferenceDedisperser2(const shared_ptr<DedispersionPlan> &plan_) :
    ReferenceDedisperserBase(plan_, 2),
    stage0_buffers(plan_),
    stage1_buffers(plan_)
{
    long S = nelts_per_segment;
    this->gpu_ringbuf = Array<float> ({plan->gmem_ringbuf_nseg * S}, af_uhost | af_zero);
    
    // Reminder: 'input_array' and 'output_arrays' are members of ReferenceDedisperserBase,
    // but are initialized by the subclass constructor.

    this->input_array = stage0_buffers.dd_bufs.at(0);   // alias
    this->output_arrays = stage1_buffers.dd_bufs;       // alias
    this->check_iobuf_shapes();
}


// Helper for ReferenceDedisperser2::_dedisperse()
// Returns segment offset in ring buffer.
static long rb_segment(const uint *rb_locs, long rb_pos, uint nelts_per_segment)
{
    uint rb_offset = rb_locs[0];  // in segments, not bytes
    uint rb_phase = rb_locs[1];   // index of (time chunk, beam) pair, relative to current pair
    uint rb_len = rb_locs[2];     // number of (time chunk, beam) pairs in ringbuf (same as Ringbuf::rb_len)
    uint rb_nseg = rb_locs[3];    // number of segments per (time chunk, beam) (same as Ringbuf::nseg_per_beam)

    uint i = (rb_pos + rb_phase) % rb_len;
    long s = rb_offset + (i * rb_nseg);

    return s;
}


void ReferenceDedisperser2::_dedisperse(long itime, long ibeam)
{
    long S = nelts_per_segment;
    long rb_pos = itime*total_beams + ibeam;
    float *rbuf = gpu_ringbuf.data;
    
    // Step 1: run LaggedDownsampler.
    // Step 2: run stage0 dedispersion kernels.
    this->stage0_buffers.apply_lagged_downsampler(ibeam);    
    this->stage0_buffers.apply_dedispersion_kernels(itime, ibeam);

    // Step 3: copy stage0 -> ringbuf.

    for (int ids = 0; ids < nds; ids++) {
	const DedispersionPlan::Stage0Tree &st0 = plan->stage0_trees.at(ids);

	long nchan0 = pow2(st0.rank0);
	long nchan1 = pow2(st0.rank1);
	long nt_ds = st0.nt_ds;
	long ns = xdiv(nt_ds, S);

	// (rb_locs0, src0) = base pointers for tree
	const uint *rb_locs0 = plan->stage0_rb_locs.data + (4 * st0.base_segment);
	const float *src0 = stage0_buffers.dd_bufs.at(ids).data;
	long src_bstride = stage0_buffers.dd_bufs.at(ids).strides[0];

	// Loop over segments in tree.
	for (long s = 0; s < ns; s++) {
	    for (long i1 = 0; i1 < nchan1; i1++) {
		for (long i0 = 0; i0 < nchan0; i0++) {
		    // (rb_locs1, src1) = base pointers for segment.
		    long iseg0 = s*nchan1*nchan0 + i1*nchan0 + i0;
		    const uint *rb_locs1 = rb_locs0 + 4*iseg0;
		    const float *src1 = src0 + (i1*nchan0+i0)*nt_ds + s*S;
		    
		    for (long b = 0; b < beams_per_batch; b++) {
			long s = rb_segment(rb_locs1, rb_pos+b, S);
			memcpy(rbuf + s*S, src1 + b*src_bstride, S * sizeof(float));
		    }
		}
	    }
	}
    }

    // Step 4: copy ringbuf -> stage1.

    for (int iout = 0; iout < nout; iout++) {
	const DedispersionPlan::Stage1Tree &st1 = plan->stage1_trees.at(iout);
	
	long nchan0 = pow2(st1.rank0);
	long nchan1 = pow2(st1.rank1_trigger);
	long nt_ds = st1.nt_ds;
	long ns = xdiv(st1.nt_ds, S);

	// (rb_locs0, dst0) = base pointers for tree
	const uint *rb_locs0 = plan->stage1_rb_locs.data + (4 * st1.base_segment);
	float *dst0 = stage1_buffers.dd_bufs.at(iout).data;
	long dst_bstride = stage1_buffers.dd_bufs.at(iout).strides[0];

	// Loop over segments in tree.
	for (long s = 0; s < ns; s++) {
	    for (long i0 = 0; i0 < nchan0; i0++) {
		for (long i1 = 0; i1 < nchan1; i1++) {
		    // (rb_locs1, dst1) = base pointers for segment.
		    long iseg1 = s*nchan1*nchan0 + i0*nchan1 + i1;
		    const uint *rb_locs1 = rb_locs0 + 4*iseg1;
		    float *dst1 = dst0 + (i1*nchan0+i0)*nt_ds + s*S;

		    for (long b = 0; b < beams_per_batch; b++) {
			long s = rb_segment(rb_locs1, rb_pos+b, S);
			memcpy(dst1 + b*dst_bstride, rbuf + s*S, S * sizeof(float));
		    }
		}
	    }
	}
    }

    // Step 5: run stage1 dedispersion kernels
    this->stage1_buffers.apply_dedispersion_kernels(itime, ibeam);    
}


// -------------------------------------------------------------------------------------------------


// Static member function
shared_ptr<ReferenceDedisperserBase> ReferenceDedisperserBase::make(const shared_ptr<DedispersionPlan> &plan_, int sophistication)
{
    if (sophistication == 0)
	return make_shared<ReferenceDedisperser0> (plan_);
    else if (sophistication == 1)
	return make_shared<ReferenceDedisperser1> (plan_);
    else if (sophistication == 2)
	return make_shared<ReferenceDedisperser2> (plan_);
    throw runtime_error("ReferenceDedisperserBase::make(): invalid value of 'sophistication' parameter");
}


}  // namespace pirate

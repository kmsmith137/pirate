#include "hip/hip_runtime.h"
#include "../include/pirate/DedispersionConfig.hpp"

#include <cstring>                   // strlen()
#include <algorithm>                 // std::sort()
#include <gputils/cuda_utils.hpp>    // CUDA_CALL()
#include <gputils/rand_utils.hpp>    // gputils::rand_*()
#include <gputils/string_utils.hpp>  // gputils::tuple_str()

#include "../include/pirate/constants.hpp"
#include "../include/pirate/internals/File.hpp"
#include "../include/pirate/internals/utils.hpp"    // check_rank(), is_empty_string()
#include "../include/pirate/internals/inlines.hpp"  // xdiv(), pow2(), print_kv()
#include "../include/pirate/internals/YamlFile.hpp"

#include <yaml-cpp/emitter.h>

using namespace std;

namespace pirate {
#if 0
}  // editor auto-indent
#endif


bool operator==(const DedispersionConfig::EarlyTrigger &x, const DedispersionConfig::EarlyTrigger &y)
{
    return (x.ds_level == y.ds_level) && (x.tree_rank == y.tree_rank);
}

bool operator>(const DedispersionConfig::EarlyTrigger &x, const DedispersionConfig::EarlyTrigger &y)
{
    if (x.ds_level > y.ds_level)
	return true;
    if (x.ds_level < y.ds_level)
	return false;
    
    if (x.tree_rank > y.tree_rank)
	return true;
    if (x.tree_rank < y.tree_rank)
	return false;
    
    return false;  // equal
}

bool operator<(const DedispersionConfig::EarlyTrigger &x, const DedispersionConfig::EarlyTrigger &y)
{
    return y > x;
}

ostream &operator<<(ostream &os, const DedispersionConfig::EarlyTrigger &et)
{
    os << "(ds=" << et.ds_level << ",rk=" << et.tree_rank << ")";
    return os;
};


// -------------------------------------------------------------------------------------------------


// Helper for DedispersionConfig::get_bytes_per_compressed_segment() and related methods.
static int dtype_size(const string &dtype)
{
    assert(!is_empty_string(dtype));
    const char *s = dtype.c_str();

    if (!strcmp(s, "float32"))
	return 4;
    else if (!strcmp(s, "float16"))
	return 2;
    else if (!strcmp(s, "int8"))
	return 1;

    stringstream ss;
    ss << "dtype_size(): unrecognized dtype '" << dtype << "'";
    throw runtime_error(ss.str());
}


int DedispersionConfig::get_uncompressed_dtype_size() const
{
    assert(!is_empty_string(this->uncompressed_dtype));
    return dtype_size(uncompressed_dtype);
}

int DedispersionConfig::get_nelts_per_segment() const
{
    assert(!is_empty_string(this->uncompressed_dtype));
    return xdiv(constants::bytes_per_segment, dtype_size(uncompressed_dtype));
}

int DedispersionConfig::get_bytes_per_compressed_segment() const
{
    assert(!is_empty_string(this->compressed_dtype));
    assert(!is_empty_string(this->uncompressed_dtype));

    int nelts = this->get_nelts_per_segment();
    int usize = dtype_size(uncompressed_dtype);
    int csize = dtype_size(compressed_dtype);

    assert(csize <= usize);
    return (nelts * csize) + ((csize < usize) ? usize : 0);
}


void DedispersionConfig::add_early_trigger(ssize_t ds_level, ssize_t tree_rank)
{
    EarlyTrigger e;
    e.ds_level = ds_level;
    e.tree_rank = tree_rank;
    this->early_triggers.push_back(e);
    
    // Incredibly lazy -- add and re-sort
    std::sort(early_triggers.begin(), early_triggers.end());
}


void DedispersionConfig::add_early_triggers(ssize_t ds_level, std::initializer_list<ssize_t> tree_ranks)
{
    for (ssize_t tree_rank: tree_ranks) {
	EarlyTrigger e;
	e.ds_level = ds_level;
	e.tree_rank = tree_rank;
	this->early_triggers.push_back(e);
    }
    
    // Incredibly lazy -- add and re-sort
    std::sort(early_triggers.begin(), early_triggers.end());
}


// Helper for DedispersionConfig::validate()
static void check_dtype(const char *name, const string &val, const std::vector<const char *> &valid_vals)
{
    if (is_empty_string(val)) {
	stringstream ss;
	ss << "DedispersionConfig::" << name << " is uninitialized";
	throw runtime_error(ss.str());
    }

    const char *vs = val.c_str();
    
    for (const char *valid_val: valid_vals)
	if (!strcmp(vs, valid_val))
	    return;

    stringstream ss2;
    ss2 << "DedispersionConfig::" << name << " '" << val << "' is invalid."
       << " Valid values are: " << gputils::tuple_str(valid_vals);
    
    throw runtime_error(ss2.str());
}

			
void DedispersionConfig::validate() const
{
    // Check that all members have been initialized.
    assert(tree_rank >= 0);
    assert(num_downsampling_levels > 0);
    assert(time_samples_per_chunk > 0);
    assert(is_sorted(early_triggers));
    assert(beams_per_gpu > 0);
    assert(beams_per_batch > 0);
    assert(num_active_batches > 0);
    assert(gmem_nbytes_per_gpu > 0);
    
    check_dtype("uncompressed_dtype", uncompressed_dtype, {"float32","float16"});
    check_dtype("compressed_dtype", compressed_dtype, {"float32","float16","int8"});
    assert(dtype_size(compressed_dtype) <= dtype_size(uncompressed_dtype));

    // GPU configuration.
    assert((beams_per_gpu % beams_per_batch) == 0);
    assert((num_active_batches * beams_per_batch) <= beams_per_gpu);

    int min_rank = (num_downsampling_levels > 1) ? 1 : 0;
    check_rank(tree_rank, "DedispersionConfig", min_rank);

    // Note: call get->nelts_per_segment() after check_dtype(uncompressed_dtype).
    int nelts_per_segment = this->get_nelts_per_segment();
    int min_nt = nelts_per_segment * pow2(num_downsampling_levels-1);
    
    if (time_samples_per_chunk % min_nt) {
	stringstream ss;
	ss << "DedispersionConfig: time_samples_per_chunk=" << time_samples_per_chunk
	   << " must be a multiple of " << min_nt
	   << " (this value depends on dtype and num_downsampling levels)";
	throw runtime_error(ss.str());
    }
    
    // Check validity of early triggers.

    int dslevel_curr = 0;
    int ntrigger_curr = 0;  // running trigger count at current downsampling level
    int ntrigger_max = 0;   // max trigger count, over all downsampling levels
    
    for (const EarlyTrigger &et: early_triggers) {
	ssize_t ds_rank = et.ds_level ? (tree_rank-1) : (tree_rank);
	ssize_t ds_rank0 = ds_rank / 2;
	
	assert((et.ds_level >= 0) && (et.ds_level < num_downsampling_levels));
	assert((et.tree_rank >= ds_rank0) && (et.tree_rank < ds_rank));

	if (et.ds_level != dslevel_curr) {
	    dslevel_curr = et.ds_level;
	    ntrigger_curr = 0;
	}

	ntrigger_curr++;
	ntrigger_max = std::max(ntrigger_max, ntrigger_curr);
    }

    if (ntrigger_max > constants::max_early_triggers_per_downsampling_level) {
	stringstream ss;
	ss << "DedispersionConfig: per-dslevel early trigger count (="
	   << ntrigger_max << ") exceeds maximum (="
	   << constants::max_early_triggers_per_downsampling_level
	   << "). You can either use fewer early triggers, or recompile pirate"
	   << "after changing max_early_triggers_per_downsampling_level in "
	   << "include/pirate/constants.hpp.";
	throw runtime_error(ss.str());
    }
}


void DedispersionConfig::print(ostream &os, int indent) const
{
    print_kv("tree_rank", tree_rank, os, indent);
    print_kv("num_downsampling_levels", num_downsampling_levels, os, indent);
    print_kv("time_samples_per_chunk", time_samples_per_chunk, os, indent);
    print_kv("uncompressed_dtype", uncompressed_dtype, os, indent);
    print_kv("compressed_dtype", compressed_dtype, os, indent);
    print_kv("early_triggers", gputils::tuple_str(early_triggers, " "), os, indent);
    
    print_kv("beams_per_gpu", beams_per_gpu, os, indent);
    print_kv("beams_per_batch", beams_per_batch, os, indent);
    print_kv("num_active_batches", num_active_batches, os, indent);
    print_kv_nbytes("gmem_nbytes_per_gpu", gmem_nbytes_per_gpu, os, indent);

    // Only print these members if they differ from default values.
    
    if (!use_hugepages)
	print_kv("use_hugepages", use_hugepages, os, indent);
    if (force_ring_buffers_to_host)
	print_kv("force_ring_buffers_to_host", force_ring_buffers_to_host, os, indent);
    if (bloat_dedispersion_plan)
	print_kv("bloat_dedispersion_plan", bloat_dedispersion_plan, os, indent);
    if (planner_verbosity > 0)
	print_kv("planner_verbosity", planner_verbosity, os, indent);
}


void DedispersionConfig::to_yaml(YAML::Emitter &emitter) const
{
    this->validate();
    
    emitter
	<< YAML::BeginMap
	<< YAML::Key << "tree_rank" << YAML::Value << tree_rank
	<< YAML::Key << "num_downsampling_levels" << YAML::Value << num_downsampling_levels
	<< YAML::Key << "time_samples_per_chunk" << YAML::Value << time_samples_per_chunk
	<< YAML::Key << "uncompressed_dtype" << YAML::Value << uncompressed_dtype
	<< YAML::Key << "compressed_dtype" << YAML::Value << compressed_dtype
	<< YAML::Key << "early_triggers"
	<< YAML::Value 
	<< YAML::BeginSeq;

    for (const auto &early_trigger: this->early_triggers) {
	emitter
	    << YAML::Flow
	    << YAML::BeginMap
	    << YAML::Key << "ds_level" << YAML::Value << early_trigger.ds_level
	    << YAML::Key << "tree_rank" << YAML::Value << early_trigger.tree_rank
	    << YAML::EndMap;
    }
    
    emitter
	<< YAML::EndSeq
	<< YAML::Key << "beams_per_gpu" << YAML::Value << beams_per_gpu
	<< YAML::Key << "beams_per_batch" << YAML::Value << beams_per_batch
	<< YAML::Key << "num_active_batches" << YAML::Value << num_active_batches
	<< YAML::Key << "gmem_nbytes_per_gpu" << YAML::Value << gmem_nbytes_per_gpu
	<< YAML::Comment(gputils::nbytes_to_str(gmem_nbytes_per_gpu))
	<< YAML::EndMap;
}


string DedispersionConfig::to_yaml_string() const
{
    YAML::Emitter emitter;
    this->to_yaml(emitter);
    return emitter.c_str();
}


void DedispersionConfig::to_yaml(const std::string &filename) const
{
    YAML::Emitter emitter;
    this->to_yaml(emitter);
    const char *s = emitter.c_str();

    File f(filename, O_WRONLY | O_CREAT | O_TRUNC);
    f.write(s, strlen(s));
}


// -------------------------------------------------------------------------------------------------


// static member function
DedispersionConfig DedispersionConfig::from_yaml(const string &filename, int verbosity)
{
    YamlFile f(filename, verbosity);
    return DedispersionConfig::from_yaml(f);
}


// static member function
DedispersionConfig DedispersionConfig::from_yaml(const YamlFile &f)
{
    DedispersionConfig ret;

    ret.tree_rank = f.get_scalar<long> ("tree_rank");
    ret.num_downsampling_levels = f.get_scalar<long> ("num_downsampling_levels");
    ret.time_samples_per_chunk = f.get_scalar<long> ("time_samples_per_chunk");
    ret.uncompressed_dtype = f.get_scalar<string> ("uncompressed_dtype");
    ret.compressed_dtype = f.get_scalar<string> ("compressed_dtype");
    ret.beams_per_gpu = f.get_scalar<long> ("beams_per_gpu");
    ret.beams_per_batch = f.get_scalar<long> ("beams_per_batch");
    ret.num_active_batches = f.get_scalar<long> ("num_active_batches");
    ret.gmem_nbytes_per_gpu = f.get_scalar<long> ("gmem_nbytes_per_gpu");

    YamlFile ets = f["early_triggers"];

    for (long i = 0; i < ets.size(); i++) {
	YamlFile et = ets[i];
	long ds_level = et.get_scalar<long> ("ds_level");
	long tree_rank = et.get_scalar<long> ("tree_rank");
	ret.add_early_trigger(ds_level, tree_rank);
	et.check_for_invalid_keys();
    }	
    
    f.check_for_invalid_keys();
    
    ret.validate();
    return ret;
}


// static member function
DedispersionConfig DedispersionConfig::make_random()
{
    DedispersionConfig ret;
    ret.num_downsampling_levels = gputils::rand_int(1, 5);

    // Ensure compressed_dtype is narrower than uncompressed_dtype.
    do {
	ret.uncompressed_dtype = gputils::rand_element({ "float32", "float16" });
	ret.compressed_dtype = gputils::rand_element({ "float32", "float16", "int8" });
    } while (dtype_size(ret.compressed_dtype) > dtype_size(ret.uncompressed_dtype));

    // Randomly choose a tree rank, but bias toward a high number.
    int max_rank = 10;
    int min_rank = (ret.num_downsampling_levels > 1) ? 1 : 0;
    double x = gputils::rand_uniform(min_rank*min_rank, (max_rank+1)*(max_rank+1));
    ret.tree_rank = int(sqrt(x));

    // Note: call ret.get_nelts_per_segment() after setting ret.uncompressed_dtype
    int max_nt_chunk = 2048;
    int min_nt_chunk = ret.get_nelts_per_segment() * pow2(ret.num_downsampling_levels-1);
    int nchunks = gputils::rand_int(1, xdiv(max_nt_chunk,min_nt_chunk)+1);
    ret.time_samples_per_chunk = min_nt_chunk * nchunks;

    // Early triggers
    
    for (int ds_level = 0; ds_level < ret.num_downsampling_levels; ds_level++) {
	int rank = ds_level ? (ret.tree_rank-1) : ret.tree_rank;;
	int min_et_rank = rank/2;
	int max_et_rank = rank-1;
	int num_candidates = max_et_rank - min_et_rank + 1;
	int max_triggers = std::min(num_candidates, constants::max_early_triggers_per_downsampling_level);

	if (max_triggers <= 0)
	    continue;

	// Randomly choose a trigger count, but bias toward a low number.
	double y = gputils::rand_uniform(-1.0, log(max_triggers+0.5));
	int num_triggers = int(exp(y));

	vector<int> et_ranks(num_candidates);
	for (int i = 0; i < num_candidates; i++)
	    et_ranks[i] = min_et_rank + i;

	gputils::randomly_permute(et_ranks);
	et_ranks.resize(num_triggers);
	std::sort(et_ranks.begin(), et_ranks.end());

	for (int et_rank: et_ranks)
	    ret.add_early_trigger(ds_level, et_rank);
    }
	
    // FIXME support these members
    ret.beams_per_gpu = 1;
    ret.beams_per_batch = 1;
    ret.num_active_batches = 1;
    ret.gmem_nbytes_per_gpu = 10L * 1000L * 1000L * 1000L;

    ret.validate();
    return ret;
}


}  // namespace pirate

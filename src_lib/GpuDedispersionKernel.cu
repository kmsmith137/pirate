#include "hip/hip_runtime.h"
#include "../include/pirate/internals/GpuDedispersionKernel.hpp"
#include "../include/pirate/internals/dedispersion_inbufs.hpp"
#include "../include/pirate/internals/dedispersion_outbufs.hpp"
#include "../include/pirate/internals/inlines.hpp"   // pow2(), is_aligned(), simd_type
#include "../include/pirate/internals/utils.hpp"     // bit_reverse_slow()
#include "../include/pirate/constants.hpp"

#include <sstream>
#include <gputils/cuda_utils.hpp>  // CUDA_CALL()

using namespace std;
using namespace gputils;

namespace pirate {
#if 0
}  // editor auto-indent
#endif


// Defined in dedispersion_kernel_implementation.hpp
// Instantiated in src_lib/template_instantiations/*.cu
template<typename T, class Inbuf, class Outbuf> extern void dedisperse_r1(typename Inbuf::device_args, typename Outbuf::device_args, T *rstate, long ntime, uint *integer_constants, long rb_pos);
template<typename T, class Inbuf, class Outbuf> extern void dedisperse_r2(typename Inbuf::device_args, typename Outbuf::device_args, T *rstate, long ntime, uint *integer_constants, long rb_pos);
template<typename T, class Inbuf, class Outbuf> extern void dedisperse_r3(typename Inbuf::device_args, typename Outbuf::device_args, T *rstate, long ntime, uint *integer_constants, long rb_pos);
template<typename T, class Inbuf, class Outbuf> extern void dedisperse_r4(typename Inbuf::device_args, typename Outbuf::device_args, T *rstate, long ntime, uint *integer_constants, long rb_pos);
template<typename T, class Inbuf, class Outbuf> extern void dedisperse_r5(typename Inbuf::device_args, typename Outbuf::device_args, T *rstate, long ntime, uint *integer_constants, long rb_pos);
template<typename T, class Inbuf, class Outbuf> extern void dedisperse_r6(typename Inbuf::device_args, typename Outbuf::device_args, T *rstate, long ntime, uint *integer_constants, long rb_pos);
template<typename T, class Inbuf, class Outbuf> extern void dedisperse_r7(typename Inbuf::device_args, typename Outbuf::device_args, T *rstate, long ntime, uint *integer_constants, long rb_pos);
template<typename T, class Inbuf, class Outbuf> extern void dedisperse_r8(typename Inbuf::device_args, typename Outbuf::device_args, T *rstate, long ntime, uint *integer_constants, long rb_pos);


template<typename T> struct _is_float32 { };
template<> struct _is_float32<float>   { static constexpr bool value = true; };
template<> struct _is_float32<__half>  { static constexpr bool value = false; };
template<> struct _is_float32<__half2> { static constexpr bool value = false; };


// When shared memory ring buffer is saved/restored in global memory, how many cache lines do we need?
template<typename T, int Rank> struct _gs_ncl { };

// Precomputed in git/chord/frb_search/r8_hacking.py
template<> struct _gs_ncl<float,5>    { static constexpr int value = 6; };
template<> struct _gs_ncl<__half2,5>  { static constexpr int value = 3; };
template<> struct _gs_ncl<float,6>    { static constexpr int value = 25; };
template<> struct _gs_ncl<__half2,6>  { static constexpr int value = 12; };
template<> struct _gs_ncl<float,7>    { static constexpr int value = 105; };
template<> struct _gs_ncl<__half2,7>  { static constexpr int value = 52; };
template<> struct _gs_ncl<float,8>    { static constexpr int value = 450; };
template<> struct _gs_ncl<__half2,8>  { static constexpr int value = 224; };

// Number of global (not shared) memory cache lines needed to store ring buffers.
static __host__ int get_gs_ncl(int rank, bool is_float32)
{
    if (rank <= 4)
	return 0;
    else if (rank == 5)
	return is_float32 ? _gs_ncl<float,5>::value : _gs_ncl<__half2,5>::value;
    else if (rank == 6)
	return is_float32 ? _gs_ncl<float,6>::value : _gs_ncl<__half2,6>::value;
    else if (rank == 7)
	return is_float32 ? _gs_ncl<float,7>::value : _gs_ncl<__half2,7>::value;
    else if (rank == 8)
	return is_float32 ? _gs_ncl<float,8>::value : _gs_ncl<__half2,8>::value;
    else
	throw runtime_error("bad arguments to get_gs_ncl()");
}

// Shared memory ring buffer footprint is larger than global memory, by 2^rank cache lines.
static __host__ int get_shmem_nbytes(int rank, bool is_float32)
{
    int ncl = pow2(rank) + get_gs_ncl(rank, is_float32);
    return ncl * 128;  // 1 cache line = 128 bytes
}


// The "integer constants" array looks like this:
//
//   uint32 control_words[2^rank1][2^rank0];  // indexed by (i,j)
//   uint32 gmem_specs[gs_ncl][2];
//
// A ring buffer "control word" consists of:
//
//   uint15 rb_base;   // base shared memory location of ring buffer (in 32-bit registers)
//   uint8  rb_pos;    // current position, satisfying 0 <= rb_pos < (rb_lag + 32)
//   uint8  rb_lag;    // ring buffer lag (in 32-bit registers), note that capacity = lag + 32.
//
// Depending on context, 'shmem_curr_pos' may point to either the end of the buffer
// (writer thread context), or be appropriately lagged (reader thread context).
//
// A "gmem spec" is a pair describing how a global memory cache line gets scattered into shared memory.
//
//   uint32 shmem_base;  // in 32-bit registers, will always be a multiple of 32
//   uint32 gap_bits;    // FIXME write comment explaining this
//
// FIXME it would probably be better to keep the integer constants array in GPU constant memory.
// (Currently we keep it in global memory.) Before doing this, I wanted to answer some initial
// questions about constant memory (search "CHORD TODO" google doc for "constant memory").


// If on_gpu=false, array is returned on host
// If on_gpu=true, array is returned on GPU.

static __host__ Array<uint> make_integer_constants(int rank, bool is_float32, bool on_gpu)
{
    if (rank <= 4)
	return Array<uint> ();

    assert(rank <= 8);
    int rank0 = rank >> 1;  // round down
    int rank1 = rank - rank0;
    
    int gs_ncl = get_gs_ncl(rank, is_float32);
    int shmem_nbytes = get_shmem_nbytes(rank, is_float32);
    
    // Total size of integer_constants array (control_words + gmem_specs)
    int ret_nelts = align_up(pow2(rank) + 2*gs_ncl, 32);
    Array<uint> ret({ret_nelts}, af_rhost | af_zero);

    // Tracks current shared memory footprint.
    uint shmem_nreg = 0;

    // Tracks current gmem spec.
    uint gs_icl = 0;    // global memory cache line index
    uint gs_ireg = 0;   // global memory register within cache line (in 0,1,...,31).
    uint gs_spos = 0;   // shared memory position (in 32-bit registers)
    uint gs_sbase = 0;  // shared memory "base" position (i.e. value at ireg=0).
    uint gs_gbits = 0;  // gap bits (initialized at ...)

    // We order the ring buffers so that all the zero-lag buffers are first, followed
    // by the nonzero-lag buffers. We implement this by running the loop twice.
    //
    // (This is necessary because the 'gap_bits' logic doesn't allow two 32-register
    //  "gaps" in a row, so we can't put zero-lag buffers between nonzero-lag buffers.)

    for (int pass = 0; pass < 2; pass++) {
	for (int i = 0; i < pow2(rank1); i++) {
	    for (int j = 0; j < pow2(rank0); j++) {
		// Ring buffer lag, in 32-bit registers.
		int ff = pow2(rank1) - i - 1;
		int dm = bit_reverse_slow(j, rank0);
		int lag = (ff*dm) >> (is_float32 ? 0 : 1);

		// Process zero-lag buffers in first pass, nonzero-lag in second pass.
		if (pass != (lag ? 1 : 0))
		    continue;

		// Ensure no overflow in control word.
		assert(shmem_nreg < 32768);  // uint15 rb_base
		assert(lag < 256);           // uint8 rb_lag
		
		// Control words are stored in global memory at "writer offset".
		// To get "reader offset", set pos=0 by applying mask 0xff007fff.
		// (See read_control_words() below.)
		
		int s = i * pow2(rank0) + j;
		ret.at({s}) = shmem_nreg | (lag << 15) | (lag << 24);  // control word
	    
		for (int l = 0; l < lag; l++) {
		    if (gs_ireg == 32) {
			// Write completed gmem_spec to 'integer_constants' array.
			assert(gs_icl < gs_ncl);
			ret.at({pow2(rank) + 2*gs_icl}) = gs_sbase;
			ret.at({pow2(rank) + 2*gs_icl+1}) = gs_gbits;
			gs_icl++;
			gs_ireg = 0;
			gs_sbase = gs_spos;
			gs_gbits = 0;
		    }

		    uint spos = shmem_nreg + l;
		    
		    if (gs_ireg == 0)
			gs_sbase = gs_spos = spos;
		    
		    if (spos == gs_spos + 32) {
			gs_gbits |= (1 << gs_ireg);
			gs_spos += 32;
		    }

		    assert(gs_spos == spos);
		    gs_ireg++;
		    gs_spos++;
		}

		// Ring buffer size = (lag + 32).
		shmem_nreg += (lag + 32);
	    }
	}
    }
    
    // After loop completes, the last gmem spec should be partially or fully complete.
    assert(gs_ireg > 0);
    assert(gs_ireg <= 32);
    assert(gs_icl == (gs_ncl-1));

    // This assert ensures that we have enough shared memory "headroom".
    int gs_smax = gs_spos + (32 - gs_ireg);
    assert(gs_smax <= shmem_nreg);
    assert(shmem_nreg*4 <= shmem_nbytes);

    // Write last gmem spec.
    ret.at({pow2(rank) + 2*gs_icl}) = gs_sbase;
    ret.at({pow2(rank) + 2*gs_icl+1}) = gs_gbits;

    return on_gpu ? ret.to_gpu() : ret;
}


// -------------------------------------------------------------------------------------------------
//
// Some helper functions which should go away, after I implement gputils::Array<void>.


// Usage: Array<float> arr = uarr_get(x, "x");   // where x is an UntypedArray
template<typename T> Array<T> uarr_get(const UntypedArray &arr, const char *arr_name);

template<> Array<float> uarr_get(const UntypedArray &uarr, const char *arr_name)
{
    bool have_float32 = (uarr.data_float32.data != nullptr);
    bool have_float16 = (uarr.data_float16.data != nullptr);

    if (!have_float32 || have_float16) {
	stringstream ss;
	ss << "GpuDedispersionKernel::launch: expected "
	   << arr_name << ".data_float32 to be nonempty, and "
	   << arr_name << ".data_float16 to be empty";
	throw runtime_error(ss.str());
    }

    return uarr.data_float32;
}

template<> Array<__half> uarr_get(const UntypedArray &uarr, const char *arr_name)
{
    bool have_float32 = (uarr.data_float32.data != nullptr);
    bool have_float16 = (uarr.data_float16.data != nullptr);

    if (have_float32 || !have_float16) {
	stringstream ss;
	ss << "GpuDedispersionKernel::launch: expected "
	   << arr_name << ".data_float32 to be empty, and "
	   << arr_name << ".data_float16 to be nonempty";
	throw runtime_error(ss.str());
    }

    return uarr.data_float16;
}


bool UntypedArray::_is_float32(const char *name) const
{
    bool have_float32 = (this->data_float32.data != nullptr);
    bool have_float16 = (this->data_float16.data != nullptr);

    if (have_float32 && !have_float16)
	return true;
    else if (!have_float32 && have_float16)
	return false;
    else if (have_float32 && have_float16)
	throw runtime_error(string(name) + " has multiple dtypes?!");
    else
	throw runtime_error(string(name) + " is empty or uninitialized");
}


UntypedArray UntypedArray::slice(int axis, int start, int stop) const
{
    UntypedArray ret;

    if (this->_is_float32("UntypedArray::slice() argument"))
	ret.data_float32 = this->data_float32.slice(axis, start, stop);
    else
	ret.data_float16 = this->data_float16.slice(axis, start, stop);

    return ret;
}


void UntypedArray::fill(const UntypedArray &x)
{
    bool dst32 = this->_is_float32("UntypedArray::fill() destination argument");
    bool src32 = x._is_float32("UntypedArray::fill() source argument");

    if (dst32 != src32)
	throw runtime_error("UntypedArray::fill(): source and destination types do not match");
    
    if (dst32)
	this->data_float32.fill(x.data_float32);
    else
	this->data_float16.fill(x.data_float16);
}


bool GpuDedispersionKernel::Params::is_float32() const
{
    // Currently only "float32" and "float16" are allowed.
    if (dtype == "float32")
	return true;
    else if (dtype == "float16")
	return false;
    else if (dtype.empty())
	throw runtime_error("GpuDedispersionKernel::Params::dtype is uninitialized (or empty string)");
    else
	throw runtime_error("GpuDedispersionKernel::Params: unrecognizd dtype '" + dtype + "' (expected 'float32' or 'float16')");
}


void GpuDedispersionKernel::Params::validate(bool on_gpu) const
{
    assert(rank >= 0);
    assert(rank <= 8);
    assert(nambient > 0);
    assert(total_beams > 0);
    assert(beams_per_kernel_launch > 0);
    assert(beams_per_kernel_launch <= constants::cuda_max_y_blocks);
    assert(ntime > 0);

    // Not really necessary, but failure probably indicates an unintentional bug.
    assert(is_power_of_two(nambient));
    
    // Currently assumed throughout the pirate code.
    assert((total_beams % beams_per_kernel_launch) == 0);

    // Currently assumed by the GPU kernels.
    int nelts_per_cache_line = is_float32() ? 32 : 64;
    assert(nelts_per_segment == nelts_per_cache_line);

    assert((ntime % nelts_per_segment) == 0);
    assert(!input_is_ringbuf || !output_is_ringbuf);
    
    if (input_is_ringbuf || output_is_ringbuf) {
	long nseg = xdiv(ntime,nelts_per_segment) * nambient * pow2(rank);
	assert(ringbuf_locations.shape_equals({ nseg, 4 }));
	assert(ringbuf_locations.is_fully_contiguous());
	assert(ringbuf_nseg > 0);
	assert(ringbuf_nseg <= UINT_MAX);

	if (on_gpu) {
	    assert(ringbuf_locations.on_gpu());
	    return;
	}

	assert(ringbuf_locations.on_host());
	
	for (long iseg = 0; iseg < nseg; iseg++) {
	    const uint *rb_locs = ringbuf_locations.data + (4*iseg);
	    long rb_offset = rb_locs[0];  // in segments, not bytes
	    // long rb_phase = rb_locs[1];   // index of (time chunk, beam) pair, relative to current pair
	    long rb_len = rb_locs[2];     // number of (time chunk, beam) pairs in ringbuf (same as Ringbuf::rb_len)
	    long rb_nseg = rb_locs[3];    // number of segments per (time chunk, beam) (same as Ringbuf::nseg_per_beam)
	    assert(rb_offset + (rb_len-1)*rb_nseg < ringbuf_nseg);
	}
    }
}


// -------------------------------------------------------------------------------------------------


template<typename T, bool Lagged>
dedispersion_simple_inbuf<T,Lagged>::device_args::device_args(const UntypedArray &in_uarr, const GpuDedispersionKernel::Params &params)
{
    // If T==float, then T32 is also 'float'.
    // If T==__half, then T32 is '__half2'.
    using T32 = typename simd32_type<T>::type;

    constexpr int elts_per_cache_line = constants::bytes_per_gpu_cache_line / 4;
    constexpr int denom = 4 / sizeof(T);
    static_assert(denom * sizeof(T) == 4);

    Array<T> in_arr = uarr_get<T> (in_uarr, "in");
    
    // Expected shape is (nbeams, nambient, pow2(rank), ntime).
    assert(in_arr.ndim == 4);
    assert(in_arr.shape[0] == params.beams_per_kernel_launch);
    assert(in_arr.shape[1] == params.nambient);
    assert(in_arr.shape[2] == pow2(params.rank));
    assert(in_arr.shape[3] == params.ntime);
    assert(in_arr.get_ncontig() >= 1);
    assert(in_arr.on_gpu());

    this->in = (T32 *) in_arr.data;
    this->beam_stride32 = xdiv(in_arr.strides[0], denom);     // 32-bit stride
    this->ambient_stride32 = xdiv(in_arr.strides[1], denom);  // 32-bit stride
    this->freq_stride32 = xdiv(in_arr.strides[2], denom);     // 32-bit stride
    this->is_downsampled = params.input_is_downsampled_tree;

    // Check alignment. Not strictly necessary, but failure would be unintentional and indicate a bug somewhere.
    assert(is_aligned(in, constants::bytes_per_gpu_cache_line));   // also checks non_NULL
    assert((beam_stride32 % elts_per_cache_line) == 0);
    assert((ambient_stride32 % elts_per_cache_line) == 0);
    assert((freq_stride32 % elts_per_cache_line) == 0);
    
    // FIXME could improve these checks, by verifying that strides are non-overlapping.
    assert(beam_stride32 != 0);
    assert(ambient_stride32 != 0);
    assert(freq_stride32 != 0);
}


template<typename T>
dedispersion_ring_outbuf<T>::device_args::device_args(const UntypedArray &in_uarr, const GpuDedispersionKernel::Params &params)
{
    // If T==float, then T32 is also 'float'.
    // If T==__half, then T32 is '__half2'.
    using T32 = typename simd32_type<T>::type;

    constexpr int denom = 4 / sizeof(T);
    static_assert(denom * sizeof(T) == 4);

    Array<T> in_arr = uarr_get<T> (in_uarr, "in");
    assert(in_arr.ndim == 1);
    assert(in_arr.shape[0] == params.ringbuf_nseg * params.nelts_per_segment);
    assert(in_arr.get_ncontig() == 1);
    assert(in_arr.on_gpu());

    Array<uint> rb_loc = params.ringbuf_locations;
    assert(rb_loc.ndim == 2);
    assert(rb_loc.shape[0] == params.nambient * pow2(params.rank) * xdiv(params.ntime, params.nelts_per_segment));
    assert(rb_loc.shape[1] == 4);
    assert(rb_loc.is_fully_contiguous());
    assert(rb_loc.on_gpu());

    this->rb_base = (T32 *) in_arr.data;
    this->rb_loc = (const uint4 *) rb_loc.data;
}


// -------------------------------------------------------------------------------------------------


// FIXME reduce cut-and-paste between Inbuf::host_args and Outbuf::host_args constructors.
template<typename T>
dedispersion_simple_outbuf<T>::device_args::device_args(const UntypedArray &out_uarr, const GpuDedispersionKernel::Params &params)
{
    // If T==float, then T32 is also 'float'.
    // If T==__half, then T32 is '__half2'.
    using T32 = typename simd32_type<T>::type;

    constexpr int elts_per_cache_line = constants::bytes_per_gpu_cache_line / 4;
    constexpr int denom = 4 / sizeof(T);
    static_assert(denom * sizeof(T) == 4);

    Array<T> out_arr = uarr_get<T> (out_uarr, "in");
    
    // Expected shape is (nbeams, nambient, pow2(rank), ntime)
    assert(out_arr.ndim == 4);
    assert(out_arr.shape[0] == params.beams_per_kernel_launch);
    assert(out_arr.shape[1] == params.nambient);
    assert(out_arr.shape[2] == pow2(params.rank));
    assert(out_arr.shape[3] == params.ntime);
    assert(out_arr.get_ncontig() >= 1);
    assert(out_arr.on_gpu());

    this->out = (T32 *) out_arr.data;
    this->beam_stride32 = xdiv(out_arr.strides[0], denom);     // 32-bit stride
    this->ambient_stride32 = xdiv(out_arr.strides[1], denom);  // 32-bit stride
    this->dm_stride32 = xdiv(out_arr.strides[2], denom);     // 32-bit stride
    
    // Check alignment. Not strictly necessary, but failure would be unintentional and indicate a bug somewhere.
    assert(is_aligned(out, constants::bytes_per_gpu_cache_line));   // also checks non-NULL
    assert((beam_stride32 % elts_per_cache_line) == 0);
    assert((ambient_stride32 % elts_per_cache_line) == 0);
    assert((dm_stride32 % elts_per_cache_line) == 0);
    
    // FIXME could improve these checks, by verifying that strides are non-overlapping.
    assert(beam_stride32 != 0);
    assert(ambient_stride32 != 0);
    assert(dm_stride32 != 0);
}


// -------------------------------------------------------------------------------------------------


template<typename T, class Inbuf, class Outbuf>
struct GpuDedispersionKernelImpl : public GpuDedispersionKernel
{    
    // If T==float, then T32 is also 'float'.
    // If T==__half, then T32 is '__half2'.
    using T32 = typename simd32_type<T>::type;

    GpuDedispersionKernelImpl(const GpuDedispersionKernel::Params &params);

    virtual void launch(const UntypedArray &in, UntypedArray &out, long itime, long ibeam, hipStream_t stream) override;

    // (inbuf, outbuf, rstate, ntime, integer_constants, rb_pos)
    void (*cuda_kernel)(typename Inbuf::device_args, typename Outbuf::device_args, T32 *, long, uint *, long) = nullptr;
};


template<typename T, class Inbuf, class Outbuf>
GpuDedispersionKernelImpl<T,Inbuf,Outbuf>::GpuDedispersionKernelImpl(const Params &params_) :
    GpuDedispersionKernel(params_)
{
    if (params.rank == 1)
	this->cuda_kernel = dedisperse_r1<T32, Inbuf, Outbuf>;
    else if (params.rank == 2)
	this->cuda_kernel = dedisperse_r2<T32, Inbuf, Outbuf>;
    else if (params.rank == 3)
	this->cuda_kernel = dedisperse_r3<T32, Inbuf, Outbuf>;
    else if (params.rank == 4)
	this->cuda_kernel = dedisperse_r4<T32, Inbuf, Outbuf>;
    else if (params.rank == 5)
	this->cuda_kernel = dedisperse_r5<T32, Inbuf, Outbuf>;
    else if (params.rank == 6)
	this->cuda_kernel = dedisperse_r6<T32, Inbuf, Outbuf>;
    else if (params.rank == 7)
	this->cuda_kernel = dedisperse_r7<T32, Inbuf, Outbuf>;
    else if (params.rank == 8)
	this->cuda_kernel = dedisperse_r8<T32, Inbuf, Outbuf>;
    else
	throw runtime_error("expected 1 <= GpuDedispersionKernel::Params::active_rank <= 8");

    // Note: this->shmem_bytes is initialized by the base class constructor.
    
    if (shmem_nbytes > 48*1024) {
        CUDA_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(
	    cuda_kernel),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            shmem_nbytes
        ));
    }
}


// virtual override
template<typename T, class Inbuf, class Outbuf>
void GpuDedispersionKernelImpl<T,Inbuf,Outbuf>::launch(const UntypedArray &in_arr, UntypedArray &out_arr, long itime, long ibeam, hipStream_t stream)
{
    typename Inbuf::device_args in(in_arr, params);
    typename Outbuf::device_args out(out_arr, params);

    // Compare (itime, ibeam) with expected values.
    assert(itime == expected_itime);
    assert(ibeam == expected_ibeam);

    // Update expected (itime, ibeam).
    expected_ibeam += params.beams_per_kernel_launch;
    assert(expected_ibeam <= params.total_beams);
    
    if (expected_ibeam == params.total_beams) {
	expected_ibeam = 0;
	expected_itime++;
    }

    Array<T> rstate = uarr_get<T> (this->persistent_state, "rstate");
    T *rp = rstate.data + (ibeam * this->state_nelts_per_beam);
    long rb_pos = itime * params.total_beams + ibeam;

    // Note: the number of beams and 'ambient' tree channels are implicitly supplied
    // to the kernel via gridDim.y, gridDim.x.
    dim3 grid_dims;
    grid_dims.x = params.nambient;
    grid_dims.y = params.beams_per_kernel_launch;
    grid_dims.z = 1;

    this->cuda_kernel
	<<< grid_dims, 32 * warps_per_threadblock, shmem_nbytes, stream >>>
	(in, out, (T32 *) rp, params.ntime, this->integer_constants.data, rb_pos);
    
    CUDA_PEEK("dedispersion kernel");
}


// -------------------------------------------------------------------------------------------------


GpuDedispersionKernel::GpuDedispersionKernel(const Params &params_) :
    params(params_)
{
    params.validate(true);    // on_gpu=true
    assert(params.rank > 0);  // FIXME define _r0 for testing
    assert(!params.input_is_ringbuf);   // FIXME
	
    // FIXME remaining code is cut-and-paste from previous API -- could use a rethink.

    bool is_float32 = params.is_float32();  // note: error-checks dtype
    int nrs_per_thread;
    
    if (params.rank == 1) {
	this->warps_per_threadblock = 1;
	nrs_per_thread = 1;
    }
    else if (params.rank == 2) {
	this->warps_per_threadblock = 1;
	nrs_per_thread = 1;
    }
    else if (params.rank == 3) {
	this->warps_per_threadblock = 1;
	nrs_per_thread = 1;
    }
    else if (params.rank == 4) {
	this->warps_per_threadblock = 1;
	nrs_per_thread = is_float32 ? 3 : 2;
    }
    else if (params.rank == 5) {
	this->warps_per_threadblock = 4;
	nrs_per_thread = 1;
    }
    else if (params.rank == 6) {
	this->warps_per_threadblock = 8;
	nrs_per_thread = is_float32 ? 2 : 1;
    }
    else if (params.rank == 7) {
	this->warps_per_threadblock = 8;
	nrs_per_thread = is_float32 ? 4 : 3;
    }
    else if (params.rank == 8) {
	this->warps_per_threadblock = 16;
	nrs_per_thread = is_float32 ? 5 : 4;
    }
    else
	throw runtime_error("GpuDedispersionKernel constructor: should never get here");
    
    long swflag = (warps_per_threadblock == 1);
    long rp_ncl = params.apply_input_residual_lags ? (pow2(params.rank) - swflag) : 0;
    long rs_ncl = warps_per_threadblock * nrs_per_thread;
    long gs_ncl = get_gs_ncl(params.rank, is_float32);
    long nelts_per_small_tree = (rs_ncl + rp_ncl + gs_ncl) * (is_float32 ? 32 : 64);
    this->state_nelts_per_beam = params.nambient * nelts_per_small_tree;
    
    if (gs_ncl > 0)
	this->shmem_nbytes = 128 * (gs_ncl + pow2(params.rank));

    if (is_float32)
	this->persistent_state.data_float32 = Array<float> ({params.total_beams, state_nelts_per_beam}, af_zero | af_gpu);    
    else
	this->persistent_state.data_float16 = Array<__half> ({params.total_beams, state_nelts_per_beam}, af_zero | af_gpu);
    
    this->integer_constants = make_integer_constants(params.rank, is_float32, true);   // on_gpu=true
}


// Static member function
shared_ptr<GpuDedispersionKernel> GpuDedispersionKernel::make(const Params &params)
{
    bool rb_in = params.input_is_ringbuf;
    bool rb_out = params.output_is_ringbuf;
    bool rlag = params.apply_input_residual_lags;
    bool is_float32 = params.is_float32();

    // Select subclass template instantiation.
    // Currently 6 cases here -- more to come.

    if (!rb_in && !rb_out && !rlag && is_float32)
	return make_shared<GpuDedispersionKernelImpl<float, dedispersion_simple_inbuf<float,false>, dedispersion_simple_outbuf<float>>> (params);
    else if (!rb_in && !rb_out && !rlag && !is_float32)
	return make_shared<GpuDedispersionKernelImpl<__half, dedispersion_simple_inbuf<__half,false>, dedispersion_simple_outbuf<__half>>> (params);
    else if (!rb_in && !rb_out && rlag && is_float32)
	return make_shared<GpuDedispersionKernelImpl<float, dedispersion_simple_inbuf<float,true>, dedispersion_simple_outbuf<float>>> (params);
    else if (!rb_in && !rb_out && rlag && !is_float32)
	return make_shared<GpuDedispersionKernelImpl<__half, dedispersion_simple_inbuf<__half,true>, dedispersion_simple_outbuf<__half>>> (params);
    else if (!rb_in && rb_out && !rlag && is_float32)
	return make_shared<GpuDedispersionKernelImpl<float, dedispersion_simple_inbuf<float,false>, dedispersion_ring_outbuf<float>>> (params);
    else if (!rb_in && rb_out && !rlag && !is_float32)
	return make_shared<GpuDedispersionKernelImpl<__half, dedispersion_simple_inbuf<__half,false>, dedispersion_ring_outbuf<__half>>> (params);
    
    throw runtime_error("GpuDedispersionKernel::make(): no suitable precompiled kernel could be found");
}


}  // namespace pirate

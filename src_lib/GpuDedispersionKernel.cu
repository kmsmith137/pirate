#include "hip/hip_runtime.h"
#include "../include/pirate/internals/GpuDedispersionKernel.hpp"
#include "../include/pirate/internals/inlines.hpp"   // pow2()
#include "../include/pirate/internals/utils.hpp"     // bit_reverse_slow()
#include "../include/pirate/constants.hpp"

#include <sstream>
#include <gputils/cuda_utils.hpp>  // CUDA_CALL()

using namespace std;
using namespace gputils;

namespace pirate {
#if 0
}  // editor auto-indent
#endif


template<typename T> struct _is_float32 { };
template<> struct _is_float32<float>   { static constexpr bool value = true; };
template<> struct _is_float32<__half>  { static constexpr bool value = false; };
template<> struct _is_float32<__half2> { static constexpr bool value = false; };


// When shared memory ring buffer is saved/restored in global memory, how many cache lines do we need?
template<typename T, int Rank> struct _gs_ncl { };

// Precomputed in git/chord/frb_search/r8_hacking.py
template<> struct _gs_ncl<float,5>    { static constexpr int value = 6; };
template<> struct _gs_ncl<__half2,5>  { static constexpr int value = 3; };
template<> struct _gs_ncl<float,6>    { static constexpr int value = 25; };
template<> struct _gs_ncl<__half2,6>  { static constexpr int value = 12; };
template<> struct _gs_ncl<float,7>    { static constexpr int value = 105; };
template<> struct _gs_ncl<__half2,7>  { static constexpr int value = 52; };
template<> struct _gs_ncl<float,8>    { static constexpr int value = 450; };
template<> struct _gs_ncl<__half2,8>  { static constexpr int value = 224; };

// Number of global (not shared) memory cache lines needed to store ring buffers.
static __host__ int get_gs_ncl(int rank, bool is_float32)
{
    if (rank <= 4)
	return 0;
    else if (rank == 5)
	return is_float32 ? _gs_ncl<float,5>::value : _gs_ncl<__half2,5>::value;
    else if (rank == 6)
	return is_float32 ? _gs_ncl<float,6>::value : _gs_ncl<__half2,6>::value;
    else if (rank == 7)
	return is_float32 ? _gs_ncl<float,7>::value : _gs_ncl<__half2,7>::value;
    else if (rank == 8)
	return is_float32 ? _gs_ncl<float,8>::value : _gs_ncl<__half2,8>::value;
    else
	throw runtime_error("bad arguments to get_gs_ncl()");
}

// Shared memory ring buffer footprint is larger than global memory, by 2^rank cache lines.
static __host__ int get_shmem_nbytes(int rank, bool is_float32)
{
    int ncl = pow2(rank) + get_gs_ncl(rank, is_float32);
    return ncl * 128;  // 1 cache line = 128 bytes
}


// The "integer constants" array looks like this:
//
//   uint32 control_words[2^rank1][2^rank0];  // indexed by (i,j)
//   uint32 gmem_specs[gs_ncl][2];
//
// A ring buffer "control word" consists of:
//
//   uint15 rb_base;   // base shared memory location of ring buffer (in 32-bit registers)
//   uint9  rb_pos;    // current position, satisfying 0 <= rb_pos < (rb_lag + 32)
//   uint8  rb_lag;    // ring buffer lag (in 32-bit registers), note that capacity = lag + 32.
//
// Depending on context, 'shmem_curr_pos' may point to either the end of the buffer
// (writer thread context), or be appropriately lagged (reader thread context).
//
// A "gmem spec" is a pair describing how a global memory cache line gets scattered into shared memory.
//
//   uint32 shmem_base;  // in 32-bit registers, will always be a multiple of 32
//   uint32 gap_bits;    // FIXME write comment explaining this
//
// FIXME it would probably be better to keep the integer constants array in GPU constant memory.
// (Currently we keep it in global memory.) Before doing this, I wanted to answer some initial
// questions about constant memory (search "CHORD TODO" google doc for "constant memory").


// If on_gpu=false, array is returned on host
// If on_gpu=true, array is returned on GPU.

static __host__ Array<uint> make_integer_constants(int rank, bool is_float32, bool on_gpu)
{
    if (rank <= 4)
	return Array<uint> ();

    assert(rank <= 8);
    int rank0 = rank >> 1;  // round down
    int rank1 = rank - rank0;
    
    int gs_ncl = get_gs_ncl(rank, is_float32);
    int shmem_nbytes = get_shmem_nbytes(rank, is_float32);
    
    // Total size of integer_constants array (control_words + gmem_specs)
    int ret_nelts = align_up(pow2(rank) + 2*gs_ncl, 32);
    Array<uint> ret({ret_nelts}, af_rhost | af_zero);

    // Tracks current shared memory footprint.
    uint shmem_nreg = 0;

    // Tracks current gmem spec.
    uint gs_icl = 0;    // global memory cache line index
    uint gs_ireg = 0;   // global memory register within cache line (in 0,1,...,31).
    uint gs_spos = 0;   // shared memory position (in 32-bit registers)
    uint gs_sbase = 0;  // shared memory "base" position (i.e. value at ireg=0).
    uint gs_gbits = 0;  // gap bits (initialized at ...)

    // We order the ring buffers so that all the zero-lag buffers are first, followed
    // by the nonzero-lag buffers. We implement this by running the loop twice.
    //
    // (This is necessary because the 'gap_bits' logic doesn't allow two 32-register
    //  "gaps" in a row, so we can't put zero-lag buffers between nonzero-lag buffers.)

    for (int pass = 0; pass < 2; pass++) {
	for (int i = 0; i < pow2(rank1); i++) {
	    for (int j = 0; j < pow2(rank0); j++) {
		// Ring buffer lag, in 32-bit registers.
		int ff = pow2(rank1) - i - 1;
		int dm = bit_reverse_slow(j, rank0);
		int lag = (ff*dm) >> (is_float32 ? 0 : 1);

		// Process zero-lag buffers in first pass, nonzero-lag in second pass.
		if (pass != (lag ? 1 : 0))
		    continue;

		// Ensure no overflow in control word.
		assert(shmem_nreg < 32768);  // uint15 rb_base
		assert(lag < 256);           // uint8 rb_lag
		
		// Control words are stored in global memory at "writer offset".
		// To get "reader offset", set pos=0 by applying mask 0xff007fff.
		// (See read_control_words() below.)
		
		int s = i * pow2(rank0) + j;
		ret.at({s}) = shmem_nreg | (lag << 15) | (lag << 24);  // control word
	    
		for (int l = 0; l < lag; l++) {
		    if (gs_ireg == 32) {
			// Write completed gmem_spec to 'integer_constants' array.
			assert(gs_icl < gs_ncl);
			ret.at({pow2(rank) + 2*gs_icl}) = gs_sbase;
			ret.at({pow2(rank) + 2*gs_icl+1}) = gs_gbits;
			gs_icl++;
			gs_ireg = 0;
			gs_sbase = gs_spos;
			gs_gbits = 0;
		    }

		    uint spos = shmem_nreg + l;
		    
		    if (gs_ireg == 0)
			gs_sbase = gs_spos = spos;
		    
		    if (spos == gs_spos + 32) {
			gs_gbits |= (1 << gs_ireg);
			gs_spos += 32;
		    }

		    assert(gs_spos == spos);
		    gs_ireg++;
		    gs_spos++;
		}

		// Ring buffer size = (lag + 32).
		shmem_nreg += (lag + 32);
	    }
	}
    }
    
    // After loop completes, the last gmem spec should be partially or fully complete.
    assert(gs_ireg > 0);
    assert(gs_ireg <= 32);
    assert(gs_icl == (gs_ncl-1));

    // This assert ensures that we have enough shared memory "headroom".
    int gs_smax = gs_spos + (32 - gs_ireg);
    assert(gs_smax <= shmem_nreg);
    assert(shmem_nreg*4 <= shmem_nbytes);

    // Write last gmem spec.
    ret.at({pow2(rank) + 2*gs_icl}) = gs_sbase;
    ret.at({pow2(rank) + 2*gs_icl+1}) = gs_gbits;

    return on_gpu ? ret.to_gpu() : ret;
}


template<typename T>
void GpuDedispersionKernel<T>::launch(T *iobuf, T *rstate,
				      long nbeams, long beam_stride,
				      long nambient, long ambient_stride,
				      long row_stride, long ntime,   // number of rows is always 2^rank
				      hipStream_t stream) const
{
    constexpr int elts_per_cache_line = constants::bytes_per_gpu_cache_line / sizeof(T);

    // Check alignment (also checks that 'iobuf' and 'rstate' are non-NULL)
    // Most of these are not strictly necessary, but failure would be unintentional and indicate a bug somewhere.
    
    assert(is_aligned(iobuf, constants::bytes_per_gpu_cache_line));
    assert(is_aligned(rstate, constants::bytes_per_gpu_cache_line));
    assert((beam_stride % elts_per_cache_line) == 0);
    assert((ambient_stride % elts_per_cache_line) == 0);
    assert((row_stride % elts_per_cache_line) == 0);
    assert((ntime % elts_per_cache_line) == 0);

    assert(ntime > 0);
    assert(nbeams > 0);
    assert(nambient > 0);
    assert(is_power_of_two(nambient));

    // Currently we only support two-stage dedispersion, where each stage has rank <= 8.
    // Therefore, we expect nambient <= 2^8. (The kernel can handle larger values, but larger values
    // would be unintentional and indicate a bug somewhere.)
    assert(nambient <= 256);

    // Required by CUDA (max allowed value of gridDims.y)
    assert(nbeams < 65536);
    
    // FIXME could improve these checks, by verifying that strides are non-overlapping.
    assert(beam_stride != 0);
    assert(ambient_stride != 0);
    assert(row_stride != 0);

    // Overflow checking.

    long max_offset = pow2(params.rank) * abs(row_stride) / 2;

    if (max_offset >= (1L << 31))
	throw runtime_error("row_stride 32-bit overflow");
    if (ntime >= (1L << 31))
	throw runtime_error("ntime 32-bit overflow");
    
    T32 *iobuf2 = reinterpret_cast<T32 *> (iobuf);
    T32 *rstate2 = reinterpret_cast<T32 *> (rstate);
    long nt_cl = ntime / elts_per_cache_line;
    bool flags = params.is_downsampled_tree ? 1 : 0;

    // Convert (T strides) to (T32 strides).
    int s = integer_log2(4 / sizeof(T));
    long beam_stride2 = beam_stride >> s;
    long ambient_stride2 = ambient_stride >> s;
    long row_stride2 = row_stride >> s;
    
    dim3 grid_dims;
    grid_dims.x = nambient;
    grid_dims.y = nbeams;
    grid_dims.z = 1;

    this->kernel
	<<< grid_dims, 32 * params.warps_per_threadblock, params.shmem_nbytes, stream >>>
        (iobuf2, rstate2, beam_stride2, ambient_stride2, row_stride2, nt_cl, this->integer_constants.data, flags);
    
    CUDA_PEEK("dedispersion kernel");
}


template<typename T>
void GpuDedispersionKernel<T>::launch(Array<T> &iobuf, Array<T> &rstate, hipStream_t stream) const
{
    if (!iobuf.on_gpu())
	throw runtime_error("GpuDedispersionKernel::launch_kernek(): iobuf array must be on GPU");
    if (!rstate.on_gpu())
	throw runtime_error("GpuDedispersionKernel::launch(): rstate array must be on GPU");
    
    // The 'iobuf' amd 'rstate' arrays must have shapes
    //   iobuf.shape = { nbeams, nambient, 2^rank, ntime }   // (beam, ambient, row, time)
    //   rstate.shape = { nbeams, nambient, state_nelts_per_small_tree }

    assert(iobuf.ndim == 4);
    assert(rstate.ndim == 3);
    assert(iobuf.shape[2] == pow2(params.rank));
    assert(rstate.shape[2] == params.state_nelts_per_small_tree);
    assert(iobuf.shape[0] == rstate.shape[0]);
    assert(iobuf.shape[1] == rstate.shape[1]);

    assert(iobuf.get_ncontig() >= 1);
    assert(rstate.is_fully_contiguous());

    // Hand off to "bare pointer" version of launch(), which will do more shape/stride
    // checking, and then launch the cuda kernel.
    
    this->launch(iobuf.data,
		 rstate.data,
		 iobuf.shape[0],    // nbeams
		 iobuf.strides[0],  // beam_stride
		 iobuf.shape[1],    // nambient
		 iobuf.strides[1],  // ambient_stride
		 iobuf.strides[2],  // row_stride,
		 iobuf.shape[3],    // ntime
		 stream);
}
    

// -------------------------------------------------------------------------------------------------
//
// Protected constructor and public factory function.


template<typename T>
GpuDedispersionKernel<T>::GpuDedispersionKernel(const Params &params_, kernel_t kernel_,
						const Array<uint> &integer_constants_) :
    params(params_), kernel(kernel_), integer_constants(integer_constants_)
{
    assert(params.rank > 0);
    assert(params.warps_per_threadblock > 0);
    assert(params.state_nelts_per_small_tree > 0);
    assert(kernel != nullptr);

    if (params.shmem_nbytes > 48*1024) {
	// FIXME: I'm asusming here that hipFuncSetAttribute() is thread-safe.
	// Should try to confirm this somehow!	
	CUDA_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(
	    kernel),
	    hipFuncAttributeMaxDynamicSharedMemorySize,
	    params.shmem_nbytes
	));
    }	
}


// Static member function.
template<typename T>
shared_ptr<GpuDedispersionKernel<T>> GpuDedispersionKernel<T>::make(int rank, bool apply_input_residual_lags, bool is_downsampled_tree)
{
    constexpr int is_float32 = _is_float32<T>::value;
    
    Params params;
    params.rank = rank;
    params.apply_input_residual_lags = apply_input_residual_lags;
    params.is_downsampled_tree = is_downsampled_tree;

    int nrs_per_thread = 0;
    kernel_t kernel = nullptr;

    // Remaining code should initialize:
    //   kernel
    //   params.warps_per_threadblock
    //   nrs_per_thread
    
    if (rank == 1) {
	kernel = apply_input_residual_lags ? dedisperse_r1<T32,true> : dedisperse_r1<T32,false>;
	params.warps_per_threadblock = 1;
	nrs_per_thread = 1;
    }
    else if (rank == 2) {
	kernel = apply_input_residual_lags ? dedisperse_r2<T32,true> : dedisperse_r2<T32,false>;
	params.warps_per_threadblock = 1;
	nrs_per_thread = 1;
    }
    else if (rank == 3) {
	kernel = apply_input_residual_lags ? dedisperse_r3<T32,true> : dedisperse_r3<T32,false>;
	params.warps_per_threadblock = 1;
	nrs_per_thread = 1;
    }
    else if (rank == 4) {
	kernel = apply_input_residual_lags ? dedisperse_r4<T32,true> : dedisperse_r4<T32,false>;
	params.warps_per_threadblock = 1;
	nrs_per_thread = is_float32 ? 3 : 2;
    }
    else if (rank == 5) {
	kernel = apply_input_residual_lags ? dedisperse_r5<T32,true> : dedisperse_r5<T32,false>;
	params.warps_per_threadblock = 4;
	nrs_per_thread = 1;
    }
    else if (rank == 6) {
	kernel = apply_input_residual_lags ? dedisperse_r6<T32,true> : dedisperse_r6<T32,false>;
	params.warps_per_threadblock = 8;
	nrs_per_thread = is_float32 ? 2 : 1;
    }
    else if (rank == 7) {
	kernel = apply_input_residual_lags ? dedisperse_r7<T32,true> : dedisperse_r7<T32,false>;
	params.warps_per_threadblock = 8;
	nrs_per_thread = is_float32 ? 4 : 3;
    }
    else if (rank == 8) {
	kernel = apply_input_residual_lags ? dedisperse_r8<T32,true> : dedisperse_r8<T32,false>;
	params.warps_per_threadblock = 16;
	nrs_per_thread = is_float32 ? 5 : 4;
    }
    else {
	stringstream ss;
	ss << "GpuDedispersionKernel::make(): rank=" << rank << " is not implemented";
	throw runtime_error(ss.str());
    }

    assert(kernel != nullptr);
    assert(nrs_per_thread > 0);
    assert(params.warps_per_threadblock > 0);

    int swflag = (params.warps_per_threadblock == 1);
    int rp_ncl = apply_input_residual_lags ? (pow2(rank) - swflag) : 0;
    int rs_ncl = params.warps_per_threadblock * nrs_per_thread;
    int gs_ncl = get_gs_ncl(rank, is_float32);

    params.state_nelts_per_small_tree = (rs_ncl + rp_ncl + gs_ncl) * (128/sizeof(T));
    
    if (gs_ncl > 0)
	params.shmem_nbytes = 128 * (gs_ncl + pow2(rank));

    Array<uint> integer_constants = make_integer_constants(rank, is_float32, true);   // on_gpu=true
	
    auto kp = new GpuDedispersionKernel(params, kernel, integer_constants);
    return shared_ptr<GpuDedispersionKernel> (kp);
}


template<typename T>
void GpuDedispersionKernel<T>::print(ostream &os, int indent) const
{
    os << Indent(indent) << "GpuDedispersionKernel<" << gputils::type_name<T>() << ">\n"
       << Indent(indent+4) << "rank = " << params.rank << "\n"
       << Indent(indent+4) << "apply_input_residual_lags = " << (params.apply_input_residual_lags ? "true" : "false") << "\n"
       << Indent(indent+4) << "is_downsampled_tree = " << (params.is_downsampled_tree ? "true" : "false") << "\n"
       << Indent(indent+4) << "state_nelts_per_small_tree = " << params.state_nelts_per_small_tree << "\n"
       << Indent(indent+4) << "warps_per_threadblock = " << params.warps_per_threadblock << "\n"
       << Indent(indent+4) << "shmem_nbytes = " << params.shmem_nbytes
       << endl;
}


#define INSTANTIATE(T) \
    template void GpuDedispersionKernel<T>::launch(T*, T*, long, long, long, long, long, long, hipStream_t) const; \
    template void GpuDedispersionKernel<T>::launch(Array<T> &, Array<T> &, hipStream_t) const; \
    template GpuDedispersionKernel<T>::GpuDedispersionKernel(const Params &, kernel_t, const Array<uint> &); \
    template shared_ptr<GpuDedispersionKernel<T>> GpuDedispersionKernel<T>::make(int, bool, bool); \
    template void GpuDedispersionKernel<T>::print(ostream &os, int indent) const

INSTANTIATE(__half);
INSTANTIATE(float);


}  // namespace pirate
